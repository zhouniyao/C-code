/*
* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
*
* NVIDIA Corporation and its licensors retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* NVIDIA Corporation is strictly prohibited.
*
* Please refer to the applicable NVIDIA end user license agreement (EULA)
* associated with this source code for terms and conditions that govern
* your use of this NVIDIA software.
*
*/


#include "../common/book.h"
#include "cpu_bitmaptest.h"    //����cpu_bitmap�ĸ���

#define DIM 1000



struct hipComplex {
	float   r;
	float   i;
	hipComplex(float a, float b) : r(a), i(b)  {}
	float magnitude2(void) { return r * r + i * i; }
	hipComplex operator*(const hipComplex& a) {
		return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}
	hipComplex operator+(const hipComplex& a) {
		return hipComplex(r + a.r, i + a.i);
	}
};

int julia(int x, int y) {
	const float scale = 1.5;
	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);

	int i = 0;
	for (i = 0; i<200; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}

	return 1;
}
///����Ϊ����Ҷ������أ�ԭģԭ�����ƹ��������øĶ�


//DataBlock���������ڸ�������֮�䴫��ͼ����ڴ�
struct DataBlock {
	unsigned char   *dev_bitmap;   //�洢ҪͿɫ������
	CPUBitmap  *bitmap;
};


void kernel(unsigned char *ptr,int ticks){
	printf("kernel:%d\n",ticks);
	for (int y = 0; y<DIM; y++) {
		for (int x = 0; x<DIM; x++) {
			int offset = x + y * DIM;
			int juliaValue = julia(x, y);
			ptr[offset * 4 + 0] = (255-ticks*10) * juliaValue;  
			ptr[offset * 4 + 1] = (0 + ticks * 15)* juliaValue;
			ptr[offset * 4 + 2] = (100 + ticks * 25)* juliaValue;  //ǰ����ptr������ɫ��Ϊ���ñ�ɫ�����ԣ�������ticks����һ��ֵ
			ptr[offset * 4 + 3] = 255;  //����͸����
		}
	}
}


//���ϱ�ɫ
void generate_frame(DataBlock *d, int ticks) {
	unsigned char *ptr =  d->dev_bitmap;  //��ȡͼ�����ص���ڴ��ַ
	kernel(ptr, ticks);  //���������ͼ��ĺ˺���
}


void cleanup(DataBlock *d) {
	free(d->dev_bitmap);   
}
int main(void) {
	DataBlock data;
	CPUBitmap bitmap(DIM, DIM,&data);
	data.bitmap = &bitmap;
	data.dev_bitmap = bitmap.get_ptr();
	//unsigned char *ptr =bitmap.get_ptr();

	//kernel(ptr,1);
	//printf("frame:%p\n", generate_frame); 
	bitmap.display_and_exit((void(*)(void*, int))generate_frame,
		(void(*)(void*))cleanup);
}

