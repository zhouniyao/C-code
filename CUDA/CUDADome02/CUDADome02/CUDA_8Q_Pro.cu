
#include <hip/hip_runtime.h>
#if 0
#include "E:\VisualStudio\CUDA\common\book.h"
#include <windows.h>
#include <stdio.h>
#include <stdlib.h>

#define N 8

#define MAXSIZE 100
//�Ϸ����жϡ�����rowĳ������֮ǰ�����бȽϣ�Q[]�洢֮ǰ�кϷ���λ��
__device__ int isLegal(int L[], int row, int pos )
{
	for(int i=0; i<row; i++)
	{
		if(L[i] == -1)return 0;
		if(pos == L[i])return 0;
		if(L[i] - pos == i - row)return 0;//���Խ���
		if(L[i] - pos == row - i)return 0;//б�Խ���
	}
	
	return 1;
}
//__device__ static int myCount = 0;//ͳ�ƽ����
__device__ void findQueen(int* L, int row, int* count)
{
	int i;
	if(row == N)
	{
		return;
	}
	for(i=0; i<N; i++)
	{
		if(isLegal(L, row, i))
		{
			L[row] = i;//�����ҵ��Ϸ�λ��
			if(row == N-1)//�����ж��������
			{
				(*count)++;
				//myCount++;
				L[row] = -1;//�������Ҹ���i���Ժ��λ��
				//copyQueen
			}
			findQueen(L, row+1, count);
		}
	}
}
__global__ void kernel(int* part_sum)//*sumÿ��Ԫ��Ϊ���ֽ����
{
	int L[N];
	int x = threadIdx.x;//�߳�����
	//__shared__ int cache[N];//�����ڴ�
	int temp = 0;
	if(x < N)
	{
		L[0] = x;
		findQueen(L, 1, &temp);	//��1-10
		printf("partJie = %d\n",temp);
	}
	part_sum[x] = temp;
	__syncthreads();
	//��Լ����
	// i = (N)/2;
	//while(i != 0)
	//{
	//	if(x <= i)
	//		cache[x] += cache[x + i];
	//	__syncthreads();
	//	i /= 2;
	//}	
}

int main()
{
	int i=0,sum=0;
	int count[N] = {0};
	int* dev_cnt;


	cudaMalloc((void**)&dev_cnt,N*sizeof(int)); 
	cudaMemcpy(dev_cnt, count,
                              N*sizeof(int),
                              cudaMemcpyHostToDevice);
	//HANDLE_ERROR( cudaMemcpy(dev_L, L,
 //                             N*sizeof(int),
 //                             cudaMemcpyHostToDevice) );
			/*ʱ���ʱ����*/
	LARGE_INTEGER t1,t2,tc;//����ʱ�����
	QueryPerformanceFrequency(&tc);//
	QueryPerformanceCounter(&t1);//

	kernel<<<1,N>>>(dev_cnt);
	cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
	
	QueryPerformanceCounter(&t2);//����ʱ��
	for(int i=0; i<N; i++)
		sum += count[i];

	printf("\n�ܹ�%d����\n", sum);

	printf("Use Time:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//��ӡ��ʱ
	cudaFree(dev_cnt);
	return 0;
}
#endif