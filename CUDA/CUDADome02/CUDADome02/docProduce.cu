
#include <hip/hip_runtime.h>
#if 0
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<windows.h>
#include "E:\VisualStudio\CUDA\common\book.h"
#define imin(a,b)  ((a)<(b)?(a):(b))
#define sum_squares(x) (x*(x+1)*(2*x+1)/6)
const int N = 3 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1)/threadsPerBlock);

__global__ void doc(float* a, float* b, float* c)//�������
{
	int x = threadIdx.x + blockIdx.x*blockDim.x;
	__shared__ float cache[threadsPerBlock];
	int cacheId = threadIdx.x;
	float temp = 0;
	while(x < N)
	{
		temp += a[x] * b[x];//��ȷ����ֹ������̲߳���
		//temp = a[x] * b[x];
		x += blockDim.x * gridDim.x;
	}
	cache[cacheId] = temp;
	//���߳̿��е��߳̽���ͬ��
	__syncthreads();

	//��Լ
	int i = blockDim.x/2;
	while(i != 0)
	{
		if(cacheId < i)
			cache[cacheId] += cache[cacheId + i];
		__syncthreads();
		i /= 2;
	}
	if(cacheId == 0)
	{c[blockIdx.x] = cache[0];}

}


int main(void)
{
	float *a, *b, c, *partial_c;
	float *dev_a, *dev_b, *dev_partial_c;
	a = new float[N];
	b = new float[N];
	partial_c = new float[blocksPerGrid];
	HANDLE_ERROR(cudaMalloc((void**)&dev_a,
							N*sizeof(float)));
	HANDLE_ERROR(cudaMalloc((void**)&dev_b,
							N*sizeof(float)));
	HANDLE_ERROR(cudaMalloc((void**)&dev_partial_c,
							blocksPerGrid*sizeof(float)));
	for(int i=0; i<N; i++)
	{
		a[i] = i;
		b[i] = 2*i;
	}
	HANDLE_ERROR(cudaMemcpy(dev_a, a, N*sizeof(float), cudaMemcpyHostToDevice));
	HANDLE_ERROR(cudaMemcpy(dev_b, b, N*sizeof(float), cudaMemcpyHostToDevice));
	doc<<<blocksPerGrid,threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);

	    HANDLE_ERROR( cudaMemcpy( partial_c, dev_partial_c,
                              blocksPerGrid*sizeof(float),
                              cudaMemcpyDeviceToHost ) );
	c = 0;
	for(int i=0; i<blocksPerGrid; i++)
		c += partial_c[i];
	printf("Does  GPU value %.6g =  %.6g\n",c,
			2*sum_squares((float)(N-1)));
	cudaFree(dev_a);
	cudaFree(dev_b);
	cudaFree(dev_partial_c);
	delete []a;
	delete []b;
	delete []partial_c;
	return 0;
}
#endif