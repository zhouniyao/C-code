
#include <hip/hip_runtime.h>
#if 0
#include <stdio.h>
#include <stdlib.h>
#include<windows.h>
#include"E:\�о���-2016��3��\CUDA\CUDA by Example.An Introduction to General-Purpose GPU Programming\cuda_by_example1\common\book.h"

#define N 4
#define M 100
//int (*QueenS)[N*M][N] = new int[N][N*M][N];
//�Ϸ����жϡ�����rowĳ������֮ǰ�����бȽϣ�Q[]�洢֮ǰ�кϷ���λ��
__device__ int isLegal(int* L, int row, int pos)
{
	int i;

	for(i=0; i<row; i++)
	{
		if(L[i] == -1)return 0;
		if(pos == L[i])	{return 0;}
		if(L[i] - pos == i - row)return 0;//���Խ���
		if(L[i] - pos == row - i)return 0;//б�Խ���
	}
	return 1;
}
//��������_������н�����ά����ĳ��
//void saveQueen(int Queen[][N], int cnt, int* dev_L)
//{	
//	int tmp[N] ={0};
//	int cnt1;
//	cudaMemcpy(tmp, dev_L, N*sizeof(int), cudaMemcpyDeviceToHost );
//	cudaMemcpy(&cnt1, &cnt, sizeof(int), cudaMemcpyDeviceToHost );
//	for(int i=0; i<N; i++)
//		Queen[cnt1][i] = tmp[i];	
//}
//��ӡ�ʺ�
void showQueen(int Queen[][N], int count)
{
	for(int i=0; i<=count; i++)
	{
		printf("��%d���⣺ ",i+1);
		for(int j=0; j<N; j++)
			printf("(%d,%d)",j,Queen[i][j]);
		printf("\n");
	}
}
__global__ void kernel(int* L, int* count)
{
	//int x = blockIdx.x;
	int x = threadIdx.x;
	L[0] = x;
	int i1, i2, i3;
	for(i1=0; i1<N; i1++ )
	{
		if(isLegal(L, 1, i1))
		{
			L[1] = i1;
			for(i2=0; i2<N; i2++)
			{
				if(isLegal(L, 2, i2))
				{
					L[2] = i2;
					for(i3=0; i3<N; i3++)
					{
						if(isLegal(L, 3, i3))
						{
							L[3] = i3;
							count[x]++;
							//saveQueen(QueenS[x], count[x], L);
						}
					}
				}
			}
		}
	}
}
int main(void)
{
	int* dev_L;
	int* dev_CNT;
	int myCount[N] = {0};
	int i,sum = 0;
	cudaMalloc((void**)&dev_CNT, M*sizeof(int));
	cudaMalloc((void**)&dev_L, N*sizeof(int));	
	//kernel<<<N, 1>>>(dev_L, dev_CNT);
	kernel<<<1, N>>>(dev_L, dev_CNT);
	cudaMemcpy(myCount, dev_CNT, N*sizeof(int), cudaMemcpyDeviceToHost );
	for(i=0; i<N; i++)
		sum += myCount[i];
	printf("sum= %d\n",sum);
}

#endif