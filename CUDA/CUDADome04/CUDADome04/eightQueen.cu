
#include <hip/hip_runtime.h>
#if 0
#include <stdio.h>
#include <stdlib.h>
#include<windows.h>
#include"E:\研究生-2016年3月\CUDA\CUDA by Example.An Introduction to General-Purpose GPU Programming\cuda_by_example1\common\book.h"

#define N 4
#define M 100
//int (*QueenS)[N*M][N] = new int[N][N*M][N];
//合法性判断——该row某列与其之前所有行比较，Q[]存储之前行合法列位置
__device__ int isLegal(int* L, int row, int pos)
{
	int i;

	for(i=0; i<row; i++)
	{
		if(L[i] == -1)return 0;
		if(pos == L[i])	{return 0;}
		if(L[i] - pos == i - row)return 0;//正对角线
		if(L[i] - pos == row - i)return 0;//斜对角线
	}
	return 1;
}
//保存数组_保存可行解至二维数组某行
//void saveQueen(int Queen[][N], int cnt, int* dev_L)
//{	
//	int tmp[N] ={0};
//	int cnt1;
//	cudaMemcpy(tmp, dev_L, N*sizeof(int), cudaMemcpyDeviceToHost );
//	cudaMemcpy(&cnt1, &cnt, sizeof(int), cudaMemcpyDeviceToHost );
//	for(int i=0; i<N; i++)
//		Queen[cnt1][i] = tmp[i];	
//}
//打印皇后
void showQueen(int Queen[][N], int count)
{
	for(int i=0; i<=count; i++)
	{
		printf("第%d个解： ",i+1);
		for(int j=0; j<N; j++)
			printf("(%d,%d)",j,Queen[i][j]);
		printf("\n");
	}
}
__global__ void kernel(int* L, int* count)
{
	//int x = blockIdx.x;
	int x = threadIdx.x;
	L[0] = x;
	int i1, i2, i3;
	for(i1=0; i1<N; i1++ )
	{
		if(isLegal(L, 1, i1))
		{
			L[1] = i1;
			for(i2=0; i2<N; i2++)
			{
				if(isLegal(L, 2, i2))
				{
					L[2] = i2;
					for(i3=0; i3<N; i3++)
					{
						if(isLegal(L, 3, i3))
						{
							L[3] = i3;
							count[x]++;
							//saveQueen(QueenS[x], count[x], L);
						}
					}
				}
			}
		}
	}
}
int main(void)
{
	int* dev_L;
	int* dev_CNT;
	int myCount[N] = {0};
	int i,sum = 0;
	cudaMalloc((void**)&dev_CNT, M*sizeof(int));
	cudaMalloc((void**)&dev_L, N*sizeof(int));	
	//kernel<<<N, 1>>>(dev_L, dev_CNT);
	kernel<<<1, N>>>(dev_L, dev_CNT);
	cudaMemcpy(myCount, dev_CNT, N*sizeof(int), cudaMemcpyDeviceToHost );
	for(i=0; i<N; i++)
		sum += myCount[i];
	printf("sum= %d\n",sum);
}

#endif