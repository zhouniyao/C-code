
#include <hip/hip_runtime.h>
#if 0
#include <stdio.h>
#include <stdlib.h>
#include<windows.h>
#include"E:\研究生-2016年3月\CUDA\CUDA by Example.An Introduction to General-Purpose GPU Programming\cuda_by_example1\common\book.h"
#define N (32*32+133)  //每个block中最大的thread数目——max threads per block
//【如果分配的线程数大于机器内部规定数，会发生什么？】
//终止核函数运行，因为GPU有完善的内存管理机制，它会强行结束所有违反内存访问规则的进程。
__global__ void add(int* dev_a, int* dev_b, int* dev_c)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;//blockDim.x保存的是线程块中每一维的线程数量,gridDim保存【线程格】中每一维的线程块的数量；gridDim是二维的，而blockDim是三维的
	//int index = blockIdx.x;//分配N个block，每个block分配一个thread
	//int index = threadIdx.x;//单个块分配N个thread
	//if换成while，可处理任意大小的适量相加
	while(index < N)
	{
		dev_c[index] = dev_a[index] + dev_b[index];
		index += gridDim.x*blockDim.x;//步长：每个线程块中的线程数量乘以线程格中线程块的数量
	}
}

int main()
{
	int i;
	int* a = (int*)malloc(N*sizeof(int));
	int* b = (int*)malloc(N*sizeof(int));
	int* c = (int*)malloc(N*sizeof(int));
	int* dev_a;
	int* dev_b;
	int* dev_c;
	HANDLE_ERROR( cudaMalloc( (void**)&dev_a,
                             N*sizeof(int) ) );
    HANDLE_ERROR( cudaMalloc( (void**)&dev_b,
                             N*sizeof(int) ) );
	HANDLE_ERROR( cudaMalloc( (void**)&dev_c,
                             N*sizeof(int) ) );
	for(i=0; i<N; i++)
	{
		a[i] = i;
		b[i] = i*i;
		printf("%d   %d\n",a[i],b[i]);
	}
	HANDLE_ERROR( cudaMemcpy( dev_a, a, N*sizeof(int),
                            cudaMemcpyHostToDevice ) ); 
	HANDLE_ERROR( cudaMemcpy( dev_b, b, N*sizeof(int),
                            cudaMemcpyHostToDevice ) ); 
	//add<<<N,1>>>(dev_a, dev_b, dev_c);
	//add<<<1,N>>>(dev_a, dev_b, dev_c);
	add<<<(N+1023)/1024,1024>>>(dev_a, dev_b, dev_c);

	HANDLE_ERROR( cudaMemcpy( c, dev_c, N*sizeof(int),
							 cudaMemcpyDeviceToHost ) ); 
	--i;
	while(i>=0)
	{
		printf("%d\t",c[i]);
		i--;
	}
	cudaFree(dev_a);
	cudaFree(dev_b);
	cudaFree(dev_c);
	return 0;
}
#endif