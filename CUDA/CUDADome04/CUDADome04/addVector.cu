
#include <hip/hip_runtime.h>
#if 0
#include <stdio.h>
#include <stdlib.h>
#include<windows.h>
#include"E:\�о���-2016��3��\CUDA\CUDA by Example.An Introduction to General-Purpose GPU Programming\cuda_by_example1\common\book.h"
#define N (32*32+133)  //ÿ��block������thread��Ŀ����max threads per block
//�����������߳������ڻ����ڲ��涨�����ᷢ��ʲô����
//��ֹ�˺������У���ΪGPU�����Ƶ��ڴ������ƣ�����ǿ�н�������Υ���ڴ���ʹ���Ľ��̡�
__global__ void add(int* dev_a, int* dev_b, int* dev_c)
{
	int index = blockIdx.x*blockDim.x + threadIdx.x;//blockDim.x��������߳̿���ÿһά���߳�����,gridDim���桾�̸߳���ÿһά���߳̿��������gridDim�Ƕ�ά�ģ���blockDim����ά��
	//int index = blockIdx.x;//����N��block��ÿ��block����һ��thread
	//int index = threadIdx.x;//���������N��thread
	//if����while���ɴ��������С���������
	while(index < N)
	{
		dev_c[index] = dev_a[index] + dev_b[index];
		index += gridDim.x*blockDim.x;//������ÿ���߳̿��е��߳����������̸߳����߳̿������
	}
}

int main()
{
	int i;
	int* a = (int*)malloc(N*sizeof(int));
	int* b = (int*)malloc(N*sizeof(int));
	int* c = (int*)malloc(N*sizeof(int));
	int* dev_a;
	int* dev_b;
	int* dev_c;
	HANDLE_ERROR( cudaMalloc( (void**)&dev_a,
                             N*sizeof(int) ) );
    HANDLE_ERROR( cudaMalloc( (void**)&dev_b,
                             N*sizeof(int) ) );
	HANDLE_ERROR( cudaMalloc( (void**)&dev_c,
                             N*sizeof(int) ) );
	for(i=0; i<N; i++)
	{
		a[i] = i;
		b[i] = i*i;
		printf("%d   %d\n",a[i],b[i]);
	}
	HANDLE_ERROR( cudaMemcpy( dev_a, a, N*sizeof(int),
                            cudaMemcpyHostToDevice ) ); 
	HANDLE_ERROR( cudaMemcpy( dev_b, b, N*sizeof(int),
                            cudaMemcpyHostToDevice ) ); 
	//add<<<N,1>>>(dev_a, dev_b, dev_c);
	//add<<<1,N>>>(dev_a, dev_b, dev_c);
	add<<<(N+1023)/1024,1024>>>(dev_a, dev_b, dev_c);

	HANDLE_ERROR( cudaMemcpy( c, dev_c, N*sizeof(int),
							 cudaMemcpyDeviceToHost ) ); 
	--i;
	while(i>=0)
	{
		printf("%d\t",c[i]);
		i--;
	}
	cudaFree(dev_a);
	cudaFree(dev_b);
	cudaFree(dev_c);
	return 0;
}
#endif