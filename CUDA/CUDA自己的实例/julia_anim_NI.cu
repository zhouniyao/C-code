#include "hip/hip_runtime.h"
#if 1

#include "../common/book.h"
#include "../common/cpu_bitmap.h"
#include "hip/hip_runtime.h"
#include "../common/cpu_anim.h"

#define DIM 1000


struct hipComplex {
    float   r;
    float   i;
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};
//__device__ int julia( int x, int y ) {
//    const float scale = 1.5;
//    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
//    float jy = scale * (float)(DIM/2 - y)/(DIM/2);
//
//    //hipComplex c(-0.8, 0.156);
//	hipComplex c(-0.8, 0.156);//ͼ��任
//    hipComplex a(jx, jy);
//
//    int i = 0;
//    for (i=0; i<200; i++) {
//        a = a * a + c;
//        if (a.magnitude2() > 1000)
//            return 0;
//    }
//
//    return 1;
//}


__device__ int julia( int x, int y,int ticks ) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

	//�Ƕ���ת
	//double radian = ticks*PI/180;
	//float jx1 = jx*cos(radian)-jy*sin(radian);
	//float jx2 = jx*sin(radian)+jy*cos(radian);


    //hipComplex c(-0.8, 0.156);
	hipComplex c(-0.8, 0.156+ticks/1000.0);//ͼ��任
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}//����Ϊ����Ҷ����

struct DataBlock {
	unsigned char   *dev_bitmap;   //�洢ҪͿɫ������
	CPUAnimBitmap  *bitmap;
};

__global__ void kernel( unsigned char *ptr,int ticks ) {
    // map from blockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;//(��)���ڸ��߳�����+�ÿ�ǰ������������߳���
    int y = threadIdx.y + blockIdx.y * blockDim.y;//(��)���ڸ��߳�����+�ÿ�ǰ������������߳���
    int offset = x + y * blockDim.x * gridDim.x;  //��άת����һά

    // now calculate the value at that position
    int juliaValue = julia( x, y ,ticks);
/*	ptr[offset * 4 + 0] = (255-ticks*20)* juliaValue;     
	ptr[offset * 4 + 1] = (100+ticks*20) *juliaValue;
	ptr[offset * 4 + 2] = (25+ticks*20) * juliaValue;	*/	

	ptr[offset*4 + 0] = (255) * juliaValue;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;	


}
void generate_frame( DataBlock *d, int ticks ) {
    dim3    blocks(DIM/8,DIM/8);
    dim3    threads(8,8);

	/*dim3    blocks(DIM/16,DIM/16);
    dim3    threads(16,16);
*/
	//dim3 grid(DIM,DIM);
    //kernel<<<grid , 1>>>( d->dev_bitmap, ticks );
	kernel<<<blocks,threads>>>(  d->dev_bitmap, ticks );

    HANDLE_ERROR( hipMemcpy( d->bitmap->get_ptr(),
                              d->dev_bitmap,
                              d->bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );
}
void cleanup( DataBlock *d ) {
    HANDLE_ERROR( hipFree( d->dev_bitmap ) ); 
}

int main( void ) {
    DataBlock   data;
    CPUAnimBitmap  bitmap( DIM, DIM, &data );
    data.bitmap = &bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&data.dev_bitmap,
                              bitmap.image_size() ) );

    bitmap.anim_and_exit( (void (*)(void*,int))generate_frame,
                            (void (*)(void*))cleanup );
}
#endif