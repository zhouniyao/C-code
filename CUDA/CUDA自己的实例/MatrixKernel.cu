
#include <hip/hip_runtime.h>
#if 0
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<windows.h>
#include"../common/book.h"
using namespace std;
#define N 32        //每个块  最大只能取32个线程，由本机硬件配置限制

//void	add(int (*a)[N],int (*b)[N],int (*c)[N]);
__global__ void add(int (*a)[N],int (*b)[N],int (*c)[N])
{
	/*既然GPU将运行核函数的N个副本，那么如何在代码中知道
	*当前正在运行的是哪个线程块？
	*无需定义blockIdx，它是一个内置变量，包含的值就是当前执行设备代码的线程块的索引。
	*
	*/
	//int j=blockIdx.x;  //计算位于这个索引处的数据，从0开始
	//int i=blockIdx.y;

	int j=threadIdx.x;
	int i=threadIdx.y;
	if(i<N&&j<N)c[i][j]=a[i][j]+b[i][j];//only "if" no "while"
	/*
	for(int i=0;i<N;i++)
	for(int j=0;j<N;j++)
	c[i][j]=a[i][j]+b[i][j];
	*/
}
int main()
{
	int a[N][N],b[N][N],c[N][N];
	int (*dev_a)[N],(*dev_b)[N],(*dev_c)[N];
	//allocate the memory on the GPU
	HANDLE_ERROR(cudaMalloc((void**)&dev_a,N*N*sizeof(int)));
	HANDLE_ERROR(cudaMalloc((void**)&dev_b,N*N*sizeof(int)));
	HANDLE_ERROR(cudaMalloc((void**)&dev_c,N*N*sizeof(int)));

	/*
	int (*a)[N]=new int[N][N];
	int (*b)[N]=new int[N][N];
	int (*c)[N]=new int[N][N];


	int a[N][N]=(int(*)[N])malloc(sizeof(int)*N*N);//错误,为什么？a是指针常量，不能做左值
	int b[N][N]=(int(*)[N])malloc(sizeof(int)*N*N);
	int c[N][N]=(int(*)[N])malloc(sizeof(int)*N*N);
	*/
	/*
	int (*a)[N]=(int(*)[N])malloc(sizeof(int)*N*N);//分配数组指针
	int (*b)[N]=(int(*)[N])malloc(sizeof(int)*N*N);
	int (*c)[N]=(int(*)[N])malloc(sizeof(int)*N*N);
	*/
	//在cpu上为数组a[]，b[]赋初始值
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			a[i][j]=i+j,b[i][j]=i*j+1;
			printf("a[%d][%d]=%4d\t b[%d][%d]=%4d  \t",i,j,a[i][j],i,j,b[i][j]);

		}
		if(i%1==0)printf("\n\n");
	}
	/*主机向设备内存传值（将a[]、b[]数组复制到GPU上）*/
	HANDLE_ERROR(cudaMemcpy(dev_a,a,N*N*sizeof(int),cudaMemcpyHostToDevice));
	HANDLE_ERROR(cudaMemcpy(dev_b,b,N*N*sizeof(int),cudaMemcpyHostToDevice));


	/*时间计时函数*/
	LARGE_INTEGER t1,t2,tc;//定义时间变量
	QueryPerformanceFrequency(&tc);//
	QueryPerformanceCounter(&t1);//


	/*此处两个参数，第一个参数表示设备在执行核函数是使用的并行线程块的数量。
	*例如kernel<<<256,1>>>表示将有256个线程块在GPU上运行。
	*/
	dim3 thread(N,N);//thread 仅是定义某个块内线程总数，最大为32,32*32 = 1024,正好是本机线程块包含最大线程数。

	//add<<<grid,1>>>(dev_a,dev_b,dev_c);
	add<<<1,thread>>>(dev_a,dev_b,dev_c);	//在GPU上将两数相加，返回
	//将数组c[]从GPU赋值到CPU
	HANDLE_ERROR(cudaMemcpy(c,dev_c,N*N*sizeof(int),cudaMemcpyDeviceToHost));


	printf("=========================================================================================");
	printf("result:\n");//显示 结果
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			printf(" c[%d][%d]=%4d\t",i,j,c[i][j]);

		}
		if(i%1==0)printf("\n\n");
	}

	QueryPerformanceCounter(&t2);//
	printf("Use Time:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//打印耗时

	/*
	free a;
	free b;
	free c;
	*/
	//释放GPU上分配的内存
	HANDLE_ERROR( cudaFree( dev_a ) );
	HANDLE_ERROR( cudaFree( dev_b ) );
	HANDLE_ERROR( cudaFree( dev_c ) );
	return 0;
}

#endif

//
//#include <stdio.h> 
//
//#include <stdlib.h> 
//
//#include <cuda_runtime.h> 
//
//
//
//#define ROWS 32 
//
//#define COLS 16 
//
//#define CHECK(res) if(res!=cudaSuccess){exit(-1);} 
//
//__global__ void Kerneltest(int **da, unsigned int rows, unsigned int cols) 
//
//{ 
//
//	unsigned int row = blockDim.y*blockIdx.y + threadIdx.y; 
//
//	unsigned int col = blockDim.x*blockIdx.x + threadIdx.x; 
//
//	if (row < rows && col < cols) 
//
//	{ 
//
//		da[row][col] = row*cols + col; 
//
//	} 
//
//} 
//
//
//
//int main(int argc, char **argv) 
//
//{ 
//
//	int **da = NULL; 
//
//	int **ha = NULL; 
//
//	int *dc = NULL; 
//
//	int *hc = NULL; 
//
//	cudaError_t res; 
//
//	int r, c; 
//
//	bool is_right=true; 
//
//
//
//	res = cudaMalloc((void**)(&da), ROWS*sizeof(int*));CHECK(res) 
//
//		res = cudaMalloc((void**)(&dc), ROWS*COLS*sizeof(int));CHECK(res) 
//
//		ha = (int**)malloc(ROWS*sizeof(int*)); 
//
//	hc = (int*)malloc(ROWS*COLS*sizeof(int)); 
//
//
//
//	for (r = 0; r < ROWS; r++) 
//
//	{ 
//
//		ha[r] = dc + r*COLS; 
//
//	} 
//
//	res = cudaMemcpy((void*)(da), (void*)(ha), ROWS*sizeof(int*), cudaMemcpyHostToDevice);CHECK(res) 
//
//		dim3 dimBlock(16,16); 
//
//	dim3 dimGrid((COLS+dimBlock.x-1)/(dimBlock.x), (ROWS+dimBlock.y-1)/(dimBlock.y)); 
//
//	Kerneltest<<<dimGrid, dimBlock>>>(da, ROWS, COLS); 
//
//	res = cudaMemcpy((void*)(hc), (void*)(dc), ROWS*COLS*sizeof(int), cudaMemcpyDeviceToHost);CHECK(res) 
//
//
//
//		for (r = 0; r < ROWS; r++) 
//
//		{ 
//
//			for (c = 0; c < COLS; c++) 
//
//			{ 
//
//				printf("%4d ", hc[r*COLS+c]); 
//
//				if (hc[r*COLS+c] != (r*COLS+c)) 
//
//				{ 
//
//					is_right = false; 
//
//				} 
//
//			} 
//
//			printf("\n"); 
//
//		} 
//
//		printf("the result is %s!\n", is_right? "right":"false"); 
//
//		cudaFree((void*)da); 
//
//		cudaFree((void*)dc); 
//
//		free(ha); 
//
//		free(hc); 
//
//		getchar(); 
//
//		return 0; 
//
//} 