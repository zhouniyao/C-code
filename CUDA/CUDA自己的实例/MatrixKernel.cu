
#include <hip/hip_runtime.h>
#if 0
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<windows.h>
#include"../common/book.h"
using namespace std;
#define N 32        //ÿ����  ���ֻ��ȡ32���̣߳��ɱ���Ӳ����������

//void	add(int (*a)[N],int (*b)[N],int (*c)[N]);
__global__ void add(int (*a)[N],int (*b)[N],int (*c)[N])
{
	/*��ȻGPU�����к˺�����N����������ô����ڴ�����֪��
	*��ǰ�������е����ĸ��߳̿飿
	*���趨��blockIdx������һ�����ñ�����������ֵ���ǵ�ǰִ���豸������߳̿��������
	*
	*/
	//int j=blockIdx.x;  //����λ����������������ݣ���0��ʼ
	//int i=blockIdx.y;

	int j=threadIdx.x;
	int i=threadIdx.y;
	if(i<N&&j<N)c[i][j]=a[i][j]+b[i][j];//only "if" no "while"
	/*
	for(int i=0;i<N;i++)
	for(int j=0;j<N;j++)
	c[i][j]=a[i][j]+b[i][j];
	*/
}
int main()
{
	int a[N][N],b[N][N],c[N][N];
	int (*dev_a)[N],(*dev_b)[N],(*dev_c)[N];
	//allocate the memory on the GPU
	HANDLE_ERROR(cudaMalloc((void**)&dev_a,N*N*sizeof(int)));
	HANDLE_ERROR(cudaMalloc((void**)&dev_b,N*N*sizeof(int)));
	HANDLE_ERROR(cudaMalloc((void**)&dev_c,N*N*sizeof(int)));

	/*
	int (*a)[N]=new int[N][N];
	int (*b)[N]=new int[N][N];
	int (*c)[N]=new int[N][N];


	int a[N][N]=(int(*)[N])malloc(sizeof(int)*N*N);//����,Ϊʲô��a��ָ�볣������������ֵ
	int b[N][N]=(int(*)[N])malloc(sizeof(int)*N*N);
	int c[N][N]=(int(*)[N])malloc(sizeof(int)*N*N);
	*/
	/*
	int (*a)[N]=(int(*)[N])malloc(sizeof(int)*N*N);//��������ָ��
	int (*b)[N]=(int(*)[N])malloc(sizeof(int)*N*N);
	int (*c)[N]=(int(*)[N])malloc(sizeof(int)*N*N);
	*/
	//��cpu��Ϊ����a[]��b[]����ʼֵ
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			a[i][j]=i+j,b[i][j]=i*j+1;
			printf("a[%d][%d]=%4d\t b[%d][%d]=%4d  \t",i,j,a[i][j],i,j,b[i][j]);

		}
		if(i%1==0)printf("\n\n");
	}
	/*�������豸�ڴ洫ֵ����a[]��b[]���鸴�Ƶ�GPU�ϣ�*/
	HANDLE_ERROR(cudaMemcpy(dev_a,a,N*N*sizeof(int),cudaMemcpyHostToDevice));
	HANDLE_ERROR(cudaMemcpy(dev_b,b,N*N*sizeof(int),cudaMemcpyHostToDevice));


	/*ʱ���ʱ����*/
	LARGE_INTEGER t1,t2,tc;//����ʱ�����
	QueryPerformanceFrequency(&tc);//
	QueryPerformanceCounter(&t1);//


	/*�˴�������������һ��������ʾ�豸��ִ�к˺�����ʹ�õĲ����߳̿��������
	*����kernel<<<256,1>>>��ʾ����256���߳̿���GPU�����С�
	*/
	dim3 thread(N,N);//thread ���Ƕ���ĳ�������߳����������Ϊ32,32*32 = 1024,�����Ǳ����߳̿��������߳�����

	//add<<<grid,1>>>(dev_a,dev_b,dev_c);
	add<<<1,thread>>>(dev_a,dev_b,dev_c);	//��GPU�Ͻ�������ӣ�����
	//������c[]��GPU��ֵ��CPU
	HANDLE_ERROR(cudaMemcpy(c,dev_c,N*N*sizeof(int),cudaMemcpyDeviceToHost));


	printf("=========================================================================================");
	printf("result:\n");//��ʾ ���
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			printf(" c[%d][%d]=%4d\t",i,j,c[i][j]);

		}
		if(i%1==0)printf("\n\n");
	}

	QueryPerformanceCounter(&t2);//
	printf("Use Time:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//��ӡ��ʱ

	/*
	free a;
	free b;
	free c;
	*/
	//�ͷ�GPU�Ϸ�����ڴ�
	HANDLE_ERROR( cudaFree( dev_a ) );
	HANDLE_ERROR( cudaFree( dev_b ) );
	HANDLE_ERROR( cudaFree( dev_c ) );
	return 0;
}

#endif

//
//#include <stdio.h> 
//
//#include <stdlib.h> 
//
//#include <cuda_runtime.h> 
//
//
//
//#define ROWS 32 
//
//#define COLS 16 
//
//#define CHECK(res) if(res!=cudaSuccess){exit(-1);} 
//
//__global__ void Kerneltest(int **da, unsigned int rows, unsigned int cols) 
//
//{ 
//
//	unsigned int row = blockDim.y*blockIdx.y + threadIdx.y; 
//
//	unsigned int col = blockDim.x*blockIdx.x + threadIdx.x; 
//
//	if (row < rows && col < cols) 
//
//	{ 
//
//		da[row][col] = row*cols + col; 
//
//	} 
//
//} 
//
//
//
//int main(int argc, char **argv) 
//
//{ 
//
//	int **da = NULL; 
//
//	int **ha = NULL; 
//
//	int *dc = NULL; 
//
//	int *hc = NULL; 
//
//	cudaError_t res; 
//
//	int r, c; 
//
//	bool is_right=true; 
//
//
//
//	res = cudaMalloc((void**)(&da), ROWS*sizeof(int*));CHECK(res) 
//
//		res = cudaMalloc((void**)(&dc), ROWS*COLS*sizeof(int));CHECK(res) 
//
//		ha = (int**)malloc(ROWS*sizeof(int*)); 
//
//	hc = (int*)malloc(ROWS*COLS*sizeof(int)); 
//
//
//
//	for (r = 0; r < ROWS; r++) 
//
//	{ 
//
//		ha[r] = dc + r*COLS; 
//
//	} 
//
//	res = cudaMemcpy((void*)(da), (void*)(ha), ROWS*sizeof(int*), cudaMemcpyHostToDevice);CHECK(res) 
//
//		dim3 dimBlock(16,16); 
//
//	dim3 dimGrid((COLS+dimBlock.x-1)/(dimBlock.x), (ROWS+dimBlock.y-1)/(dimBlock.y)); 
//
//	Kerneltest<<<dimGrid, dimBlock>>>(da, ROWS, COLS); 
//
//	res = cudaMemcpy((void*)(hc), (void*)(dc), ROWS*COLS*sizeof(int), cudaMemcpyDeviceToHost);CHECK(res) 
//
//
//
//		for (r = 0; r < ROWS; r++) 
//
//		{ 
//
//			for (c = 0; c < COLS; c++) 
//
//			{ 
//
//				printf("%4d ", hc[r*COLS+c]); 
//
//				if (hc[r*COLS+c] != (r*COLS+c)) 
//
//				{ 
//
//					is_right = false; 
//
//				} 
//
//			} 
//
//			printf("\n"); 
//
//		} 
//
//		printf("the result is %s!\n", is_right? "right":"false"); 
//
//		cudaFree((void*)da); 
//
//		cudaFree((void*)dc); 
//
//		free(ha); 
//
//		free(hc); 
//
//		getchar(); 
//
//		return 0; 
//
//} 