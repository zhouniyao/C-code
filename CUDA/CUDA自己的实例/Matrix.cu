
#include <hip/hip_runtime.h>
#if 0
#include<iostream>
#include<stdio.h>
#include<stdlib.h>
#include<windows.h>
using namespace std;
#define N 100

void	add(int (*a)[N],int (*b)[N],int (*c)[N]);
//void	add(int (&a)[N][N],int (&b)[N][N],int (&c)[N][N]);//二维数组引用

int main()
{
	int a[N][N],b[N][N],c[N][N];
	
	//
	//int (*a)[N]=new int[N][N];
	//int (*b)[N]=new int[N][N];
	//int (*c)[N]=new int[N][N];
	
	//
	//int a[N][N]=(int(*)[N])malloc(sizeof(int)*N*N);//错误,为什么？a是指针常量，不能做左值
	//int b[N][N]=(int(*)[N])malloc(sizeof(int)*N*N);
	//int c[N][N]=(int(*)[N])malloc(sizeof(int)*N*N);
	//
	
	//int (*a)[N]=(int(*)[N])malloc(sizeof(int)*N*N);//分配给数组指针，正确
	//int (*b)[N]=(int(*)[N])malloc(sizeof(int)*N*N);
	//int (*c)[N]=(int(*)[N])malloc(sizeof(int)*N*N);
	
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			a[i][j]=i+j,b[i][j]=i*j;
			printf("a[%d][%d]=%4d\t b[%d][%d]=%4d  \t",i,j,a[i][j],i,j,b[i][j]);
			
		}
		if(i%1==0)printf("\n\n");
	}
	/*时间计时函数*/
	LARGE_INTEGER t1,t2,tc;//定义时间变量
	QueryPerformanceFrequency(&tc);//
	QueryPerformanceCounter(&t1);//
	
	add(a,b,c);
	
	
	printf("=========================================================================================\n");
	printf("result:\n");
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			printf(" c[%d][%d]=%4d\t",i,j,c[i][j]);
			
		}
		if(i%1==0)printf("\n\n");
	}

	QueryPerformanceCounter(&t2);//
	printf("Use Time:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//打印耗时

	/*
	free a;
	free b;
	free c;
	*/
}

//void	add(int (&a)[N][N],int (&b)[N][N],int (&c)[N][N])//二维数组的引用
//{
//		for(int i=0;i<N;i++)
//			for(int j=0;j<N;j++)
//				c[i][j]=a[i][j]+b[i][j];
//}
void	add(int (*a)[N],int (*b)[N],int (*c)[N])  //二维数组指针
{
		for(int i=0;i<N;i++)
			for(int j=0;j<N;j++)//一下三者等价
				c[i][j]=a[i][j]+b[i][j]; 
				//*(c[i]+j) = *(a[i]+j) + *(b[i]+j);
				//*(*(c+i)+j) = *(*(a+i)+j) + *(*(b+i)+j);
}

#endif