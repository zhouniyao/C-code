#include"E:\VisualStudio\CUDA\CUDAColorToGray\CUDAColorToGray\bmploader_zhang.h"
#include"E:\VisualStudio\CUDA\CUDAColorToGray\CUDAColorToGray\color_fading.h"
#include"E:\VisualStudio\CUDA\cudaDome03\common/cpu_bitmap.h"
#include<iostream>
using namespace std;
#define WIDTH 640
#define	HEIGHT 480

int width = WIDTH;
int height = HEIGHT;
extern void fade( unsigned char *ptr );//��ɫ

void blur(unsigned char *ptr, unsigned char *src)//ģ��
{
	int offset = 0, i = 0, count = 0;		
	for (int y=0; y<height; y++, count++) {
		for (int x=0; x<width; x++,i+=4) {
			offset = x + y * width + width * count;  
			//int left = offset - 1;
			//int right = offset + 1;
			//if (x == 0)   left++;
			//if (x == width-1) right--; 

			//int top = offset - width;
			//int bottom = offset + width;
			//if (y == 0)   top += width;
			//if (y == height-1) bottom -= width;

		/*	int left = offset - 1;
			int right = offset + 1;
			int top = offset - width;
			int topl = top - 5;
			int topr = top + 5;
			int bottom = offset + width;
			int bottoml = bottom - 5;
			int bottomr = bottom + 5;
			if (x == 0)   
			{
				left++;
				topl = top;
				bottoml = bottom;
			}
			if (x == width-1)
			{
				right--; 
				topr = top;
				bottomr = bottom;
			}

	
			if (y == 0)   
			{
				top += width;
				topr = topl = top;
			}
			if (y == height-1) 
			{
				bottom -= width;
				bottomr = bottoml = bottom;
			}
*/
			int left = i - 1;
			int right = i + 1;
			int top = i - width;
			int topl = top - 5;
			int topr = top + 5;
			int bottom = i + width;
			int bottoml = bottom - 5;
			int bottomr = bottom + 5;
			if (x == 0)   
			{
				left++;
				topl = top;
				bottoml = bottom;
			}
			if (x == width-1)
			{
				right--; 
				topr = top;
				bottomr = bottom;
			}	
			if (y == 0)   
			{
				top += width;
				topr = topl = top;
			}
			if (y == height-1) 
			{
				bottom -= width;
				bottomr = bottoml = bottom;
			}

			//��˹ģ��
			ptr[offset*4 + 0] = src[topl+0]*0.0947416+src[top+0]*0.118318+src[topr+0]*0.0947416+src[bottoml+0]*0.0947416+src[bottom+0]*0.118318+src[bottomr+0]*0.0947416+src[left+0]*0.118318+src[right+0]*0.118318;
			ptr[offset*4 + 1] = src[topl+1]*0.0947416+src[top+1]*0.118318+src[topr+1]*0.0947416+src[bottoml+1]*0.0947416+src[bottom+1]*0.118318+src[bottomr+1]*0.0947416+src[left+1]*0.118318+src[right+1]*0.118318;
			ptr[offset*4 + 2] = src[topl+2]*0.0947416+src[top+2]*0.118318+src[topr+2]*0.0947416+src[bottoml+2]*0.0947416+src[bottom+2]*0.118318+src[bottomr+2]*0.0947416+src[left+2]*0.118318+src[right+2]*0.118318;
			ptr[offset*4 + 3] = 255;

			//ptr[offset*4 + 0] = ptr[topl+0]*0.0947416+ptr[top+0]*0.118318+ptr[topr+0]*0.0947416+ptr[bottoml+0]*0.0947416+ptr[bottom+0]*0.118318+ptr[bottomr+0]*0.0947416+ptr[left+0]*0.118318+ptr[right+0]*0.118318;
			//ptr[offset*4 + 1] = ptr[topl+1]*0.0947416+ptr[top+1]*0.118318+ptr[topr+1]*0.0947416+ptr[bottoml+1]*0.0947416+ptr[bottom+1]*0.118318+ptr[bottomr+1]*0.0947416+ptr[left+1]*0.118318+ptr[right+1]*0.118318;
			//ptr[offset*4 + 2] = ptr[topl+2]*0.0947416+ptr[top+2]*0.118318+ptr[topr+2]*0.0947416+ptr[bottoml+2]*0.0947416+ptr[bottom+2]*0.118318+ptr[bottomr+2]*0.0947416+ptr[left+2]*0.118318+ptr[right+2]*0.118318;
			//ptr[offset*4 + 3] = 255;



		}
	}


	//��ʾԭͼ
	offset = 0, i = 0, count = 1;		
	for (int y=0; y<height; y++, count++) {
		for (int x=0; x<width; x++,i+=4) {
			offset = x + y * width + width * count;  
			ptr[offset*4 + 0] = src[i + 0] ;
			ptr[offset*4 + 1] = src[i + 1] ;
			ptr[offset*4 + 2] = src[i + 2] ;
			ptr[offset*4 + 3] = 255;

		}
	}

	//for (int y=0; y<height; y++, count++) {
	//	for (int x=0; x<width; x++,i+=4) {
	//		offset = x + y * width + width * count;  


	//		int left = offset - 1;
	//		int right = offset + 1;
	//		if (x == 0)   left++;
	//		if (x == width-1) right--; 

	//		int top = offset - width;
	//		int bottom = offset + width;
	//		if (y == 0)   top += width;
	//		if (y == height-1) bottom -= width;

	//		ptr[offset*4 + 0] = src[offset*4+0];
	//		ptr[offset*4 + 1] = src[offset*4+1];
	//		ptr[offset*4 + 2] = src[offset*4+2] ;
	//		ptr[offset*4 + 3] = 255;

	//		//ptr[offset*4 + 0] = (src[top+0]+src[bottom+0]+src[left+0]+src[right+0])/3;
	//		//ptr[offset*4 + 1] = (src[top+1]+src[bottom+1]+src[left+1]+src[right+1])/3;
	//		//ptr[offset*4 + 2] = (src[top+2]+src[bottom+2]+src[left+2]+src[right+2])/3 ;
	//		//ptr[offset*4 + 3] = 255;

	//		//ptr[offset*4 + 0] = (ptr[top+0]+ptr[bottom+0]+ptr[left+0]+ptr[right+0])/3;
	//		//ptr[offset*4 + 1] = (ptr[top+1]+ptr[bottom+1]+ptr[left+1]+ptr[right+1])/3;
	//		//ptr[offset*4 + 2] = (ptr[top+2]+ptr[bottom+2]+ptr[left+2]+ptr[right+2])/3 ;
	//		//ptr[offset*4 + 3] = 255;
	//	}
	//}

}

//int main()
//{
//	unsigned char *dst1;
//	dst1=(unsigned char *)malloc(sizeof(int));
//	
//	int w,h;
//	w=width;h=height;	
//	char *name="nature_monte.bmp";
//	LoadBMPFile(&dst1, &w,  &h, name);	 
//	CPUBitmap bitmap(2*w, h);
//	unsigned char *ptr = bitmap.get_ptr(); 
//	blur(ptr,dst1);
//	bitmap.display_and_exit(); 
//	free(dst1);
//
//}
