#include "hip/hip_runtime.h"

#include "E:\VisualStudio\CUDA\common\book.h"

#define  N  128



__global__ void kernel(int *A, int *C)
{
	int i = blockIdx.x;
	int j = blockIdx.y;
	if (A[i]>A[j])
		atomicAdd(&C[i], 1);
	if ((A[i] == A[j])&&(i>j))
		atomicAdd(&C[i], 1);
}
int main(void)
{
	int  A[128] = { 637, 84, 96, 2, 67, 91, 68, 355, 91, 96, 19, 56, 90, 48, 657, 99, 94, 89, 93, 76, 102, 784, 11, 12, 98, 106, 42, 112, 999, 114, 87, 21, 22, 64, 91, 632, 60, 25, 97, 74, 42, 40, 281, 37, 30, 45, 69, 67, 41, 924, 42, 40, 85, 108, 12, 22, 465, 66, 99, 118, 50, 5, 15, 49, 8, 66, 71, 38, 27, 50, 70, 77, 108, 44, 54, 26, 49, 61, 110, 16, 85, 54, 15, 129, 110, 50, 100, 27, 56, 47, 62, 2, 107, 19, 29, 20, 7, 64, 69, 26, 38, 18, 81, 67, 20, 78, 51, 40, 76, 120, 32, 128, 36, 34, 63, 128, 6, 88, 65, 108, 102, 105, 13, 33, 92, 21, 102, 20 };
	int   *B, *C;

	int *dev_A, *dev_C;
	// allocate the memory on the CPU
	B = (int*)malloc(N * sizeof(int));
	C = (int*)malloc(N * sizeof(int));
	for (int i = 0; i<N; i++)
	{
		 
		B[i] = 0;
		C[i] = 0;
	}
	// allocate the memory on the GPU
	HANDLE_ERROR(hipMalloc((void**)&dev_A, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_C, N * sizeof(int)));


	// copy the arrays 'A' and 'C' to the GPU
	HANDLE_ERROR(hipMemcpy(dev_A, A, N * sizeof(int),
		hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_C, C, N * sizeof(int),
		hipMemcpyHostToDevice));
	
	dim3   blocks(N,N);
	// capture the start time
	hipEvent_t     start, stop;
	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));
	HANDLE_ERROR(hipEventRecord(start, 0));

	kernel <<< blocks, 1 >>>(dev_A, dev_C);

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	float   elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
		start, stop));
	printf("Time to generate:  %3.1f ms\n", elapsedTime);
	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));
	
	// copy the array 'C' back from the GPU to the CPU
	HANDLE_ERROR(hipMemcpy(C, dev_C, N * sizeof(int),
		hipMemcpyDeviceToHost));
	printf("Have sorted:\n");
	for (int i = 0; i<N; i++)
	{
		B[C[i]] = A[i];
		printf("%d,", B[i]);

	}

	// free the memory we allocated on the GPU
	HANDLE_ERROR(hipFree(dev_A));
	HANDLE_ERROR(hipFree(dev_C));

	// free the memory we allocated on the CPU

	free(B);
	free(C);

	return 0;

}
