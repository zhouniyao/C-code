
#include <hip/hip_runtime.h>
//#include"E:\VisualStudio\CUDA\CUDAColorToGray\CUDAColorToGray\bmploader_zhang.h"
//#include"E:\VisualStudio\CUDA\CUDAColorToGray\CUDAColorToGray\color_fading.h"
//#include"E:\VisualStudio\CUDA\cudaDome03\common\cpu_bitmap.h"
//#include <stdio.h>
//#include <stdlib.h>
//#include <cmath>
//#define  width   1024
//#define  height  683
//#define  size       width*height
//void kernel(unsigned char *ptr, unsigned char *dst1){
//
//	for (int y = 0; y<height; y++) {
//		for (int x = 0; x<width; x++) {
//			int offset = x + y * width;
//			ptr[offset * 4 + 0] = dst1[offset * 4 + 0] * 0.299 + dst1[offset * 4 + 1] * 0.587 + dst1[offset * 4 + 2] * 0.114;
//			ptr[offset * 4 + 1] = dst1[offset * 4 + 0] * 0.299 + dst1[offset * 4 + 1] * 0.587 + dst1[offset * 4 + 2] * 0.114;
//			ptr[offset * 4 + 2] = dst1[offset * 4 + 0] * 0.299 + dst1[offset * 4 + 1] * 0.587 + dst1[offset * 4 + 2] * 0.114;
//			ptr[offset * 4 + 3] = 255;
//		}
//	}
//	 
//}
//int main(void)
//{
//	unsigned char *dst1;
//	dst1 = (unsigned char *)malloc(sizeof(int));
//	unsigned char **dst;
//	dst = &dst1;
//	int w, h;
//	w = width; h = height;
//	char *name = "un.bmp";
//	LoadBMPFile(dst, &w, &h, name);
//	CPUBitmap bitmap(w, h);
//	unsigned char *ptr = bitmap.get_ptr();
//	kernel(ptr, dst1);//��ɫ
//	unsigned char *  buffer;
//	buffer = (unsigned char *)malloc(sizeof(unsigned char )*size);
//	 
//	for (int y = 0; y<height; y++) {
//		for (int x = 0; x<width; x++) {
//			int offset = x + y * width;		
//		   buffer[offset] = ptr[offset * 4 + 0];  //  
//			 
//		}
//	}
//	int  histo[256];//ֱ��ͼ
//	for (int i = 0; i < 256; i++)
//		histo[i] = 0;
//	for (long i = 0; i < size; i++)
//		histo[buffer[i]]++;
//	/*for (int i = 0; i < 256; i++)
//	{
//		printf("%d:,%d", i, histo[i]);
//		printf("\n");
//	}*/
//	long  histoCount[256];
//	histoCount[0] = histo[0];
//	printf("%d", histoCount[0]); printf("\n");
//	for (int i = 1; i < 256; i++)
//	{
//		histoCount[i] = histo[i] + histoCount[i - 1];
//		 
//	}
//	
//	 
//	long  new_grey[256];
//	for (int i = 0; i < 256; i++) new_grey[i] = 0;
//	for (int i = 1; i < 256; i++)
//	{
//		new_grey[i] =round( (long  double)(histoCount[i] - histoCount[0]) / (size - histoCount[0]) * 255);
//		 
//	}
//	kernel(ptr, dst1);
//	for (int y = 0; y < height; y++) {
//		for (int x = 0; x < width; x++) {
//			int offset = x + y * width;
//			ptr[offset * 4 + 0] = new_grey[ptr[offset * 4 + 0]];
//			ptr[offset * 4 + 1] = new_grey[ptr[offset * 4 + 1]];
//			ptr[offset * 4 + 2] = new_grey[ptr[offset * 4 + 2]];
//			ptr[offset * 4 + 3] = 255;
//		}
//	}
//	bitmap.display_and_exit();
//	free(dst1);
//	free(buffer);	 
//
//}
