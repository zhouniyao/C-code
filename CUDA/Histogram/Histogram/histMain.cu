#include "hip/hip_runtime.h"
#include"E:\VisualStudio\CUDA\CUDAColorToGray\CUDAColorToGray\bmploader_zhang.h"
#include"E:\VisualStudio\CUDA\CUDAColorToGray\CUDAColorToGray\color_fading.h"
#include"E:\VisualStudio\CUDA\cudaDome03\common\cpu_bitmap.h"
#include "math.h"
#include<iostream>
using namespace std;
#define WIDTH 1024
#define	HEIGHT 683
int width = WIDTH;
int height = HEIGHT;

unsigned char *fade(unsigned char *ptr, unsigned char *dst)//��ɫ
{
	
	for(int y=0; y<HEIGHT; y++)
		for(int x=0; x<WIDTH; x++)
		{
			int offset = x + y * WIDTH;
			//int offset = x * WIDTH + y ;  //����
			//��RGBɫתΪ�Ҷȹ�ʽ��
			//Gray = R*0.299 + G*0.587 + B*0.114
            //ptr[offset*4 + 0] = ptr[offset*4 + 0]*0.299 + ptr[offset*4 + 1] *0.587 + ptr[offset*4 + 2] * 0.114 ;
            //ptr[offset*4 + 1] = ptr[offset*4 + 0]*0.299 + ptr[offset*4 + 1] *0.587 + ptr[offset*4 + 2] * 0.114 ;
            //ptr[offset*4 + 2] = ptr[offset*4 + 0]*0.299 + ptr[offset*4 + 1] *0.587 + ptr[offset*4 + 2] * 0.114 ;
			ptr[offset*4 + 0] = (dst[offset*4 + 0] + dst[offset*4 + 1]  + dst[offset*4 + 2] )/3 ;
            ptr[offset*4 + 1] = (dst[offset*4 + 0] + dst[offset*4 + 1]  + dst[offset*4 + 2] )/3 ;
            ptr[offset*4 + 2] = (dst[offset*4 + 0] + dst[offset*4 + 1]  + dst[offset*4 + 2] )/3 ;
            ptr[offset*4 + 3] = 255;
        }
	return ptr;		
}
void scan(unsigned char *ptr, int *hist)//ɨ�裬������ֱ��ͼ
{
	for(int y=0; y<HEIGHT; y++)
		for(int x=0; x<WIDTH; x+=4)
		{
			long int offset = x + y * WIDTH;
			//int offset = x * WIDTH + y ;  //����
			hist[ptr[offset]]++;
        }
}
//�ۻ��ֲ�����
void accumulate(int *hist)
{
	int temp[256]={0};//���ֺ�ȫ����ʼ��0
	for(int i=1;i<256;i++)
	{
		//for(int j=0;j<=i;j++)
		//	temp[i] += hist[j];
		temp[i] = temp[i-1] + hist[i];
	}
	for(int i=1;i<256;i++)
		hist[i]=temp[i];

}

//���⻯
int *balance( int *hist)
{
	int *bala = new int[256];//��¼���⻯��ĻҶ�ֵ	
	memset(bala,'\0',256*4);//ȫ����0,��ȷ������ת�����ַ��������е�����
	int i=0;//���ҵ�һ����Ϊ0�Ĳ��ֺ͵�����
	for(;i<256;i++)
		if(hist[i])break;
	int min = i;
	cout<<min<<"********************************************************************";
	for(i=min;i<256;i++)
	{
		//bala[i] = (int)((hist[i]-hist[min])*255/(width*height-hist[min])+0.5);
		bala[i] = round( (double)(hist[i] - hist[min]) * 255 / (width*height - hist[min]));
	}
	//for(i=1;i<256;i++)
	//{
	//	bala[i] = round( (double)(hist[i] - hist[0]) * 255 / (width*height - hist[0]));
	//}
	return bala;
}
void change(unsigned char *ptr, int *bala)//����ֱ�����⻯ͼ
{
	//for(int y=0; y<HEIGHT; y++)
	//	for(int x=0; x<WIDTH; x++)
	//	{
	//		int offset = x + y * WIDTH;
	//		ptr[offset*4 ] = (unsigned char)bala[ptr[offset*4 ]];
 //       }
	for(int y=0; y<HEIGHT; y++)
		for(int x=0; x<WIDTH; x++)
		{
			int offset = x + y * WIDTH;//����ÿ�����ص�
			ptr[offset*4 + 0] = bala[ptr[offset*4 + 0]];
            ptr[offset*4 + 1] = bala[ptr[offset*4 + 1]];
            ptr[offset*4 + 2] = bala[ptr[offset*4 + 2]];
            ptr[offset*4 + 3] = 255;
        }

}
int main()
{
	int w,h;
	w=width;h=height;
	unsigned char *dst;
	//dst=(unsigned char *)malloc(sizeof(char)*w*h*4);//�������ռ�
	
	CPUBitmap bitmap(w, h);
	//char *name="nature_monte.bmp";
	char *name="un.bmp";
	//LoadBMPFile(&dst, &w,  &h, name);	 
	LoadBMPFile(&bitmap.pixels, &w,  &h, name);//����ɹ�
	
	//blur(ptr,dst1);//ģ��
	//fade(bitmap.pixels, dst);//��ɫ


	//����ռ䡪��ֱ��ͼ
	int *hist = new int[256];
	memset(hist,'\0',256*4);//ȫ����0,��ȷ������ת�����ַ��������е�����

	scan(bitmap.pixels, hist);//��¼ֱ��ͼ
	accumulate(hist);//�ۻ��ֲ�����
	int *bala = balance( hist);
	change(bitmap.pixels, bala);

	bitmap.display_and_exit(); 
	free(dst);
	delete []hist;
	delete []bala;

}