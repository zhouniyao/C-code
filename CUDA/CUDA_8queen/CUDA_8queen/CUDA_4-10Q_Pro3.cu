#include "hip/hip_runtime.h"
#if 1
/*
*N�ʺ�����CUDA����ͳ�Ĵ����㷨�����ݹ���ݣ����л����滮���3��
*
*
*/
#include "E:\VisualStudio\CUDA\common\book.h"
#include <windows.h>
#include <stdio.h>
#include <stdlib.h>

#define N 8

#define MAXSIZE 100 
//�Ϸ����жϡ�����rowĳ������֮ǰ�����бȽϣ�Q[]�洢֮ǰ�кϷ���λ��
__device__ int isLegal(int* L, int row, int pos )
{
	for(int i=0; i<row; i++)
	{
		if(L[i] == -1)return 0;
		if(pos == L[i])return 0;
		if(L[i] - pos == i - row)return 0;//���Խ���
		if(L[i] - pos == row - i)return 0;//б�Խ���
	}
	
	return 1;
}
//__device__ static int myCount = 0;//ͳ�ƽ����
__device__ void findQueen(int* L, int row, int* count)
{
	int i;
	if(row == N)
	{
		return;
	}
	for(i=0; i<N; i++)
	{
		if(isLegal(L, row, i))
		{
			L[row] = i;//�����ҵ��Ϸ�λ��
			if(row == N-1)//�ݹ鵽���һ��
			{
				(*count)++;
				//myCount++;
				L[row] = -1;//�������Ҹ���i���Ժ��λ��
				//copyQueen
			}
			findQueen(L, row+1, count);//�ݹ���һ��
		}
	}
}
__device__ void Allocate(int* pa, int* pb, int* pc, int* count)
{
	int L[N] = {0};
	L[0] = *pa;
	L[1] = *pb;
	L[2] = *pc;
	findQueen(L, 3, count);
	
}

__global__ void kernel(int* part_sum, int* L)//*sumÿ��Ԫ��Ϊ���ֽ����
{
	int z = threadIdx.z;
	int y = threadIdx.y;//��������
	int x = threadIdx.x;//�߳�������
	
	int a = y;
	int b = x;
	int c = z;
	
	//__shared__ int cache[N];//�����ڴ�
	int tid = x + y*blockDim.x + z*blockDim.x*blockDim.x;
	//printf("(%d,%d,%d)\t",x,y,z);
	//printf("a,b,c = (%d, %d, %d)\n",a,b,c);
	int temp = 0;
	//if(tid < N*N)
	{		
		//if(a != b && a-b != 1 && b-a != 1)
		{	
			if((a != b && a-b != 1 && b-a != 1)&&(c != b && c-b != 1 && b-c != 1)&&(a != c && a-c != 2 && c-a != 2))
				Allocate(&a, &b, &c, &temp);
		}
		//printf("Temp = %d\n",temp);
		part_sum[tid] = temp;
	}
	//printf("PartSUM = %d\n",part_sum[tid]);
	//printf("Tid = %d\t",tid);
	//__syncthreads();
	//��Լ����
	// i = (N)/2;
	//while(i != 0)
	//{
	//	if(x <= i)
	//		cache[x] += cache[x + i];
	//	__syncthreads();
	//	i /= 2;
	//}	
}

int main()
{
	LARGE_INTEGER t1,t2,tc,t3,t4,tq;//����ʱ�����
	QueryPerformanceFrequency(&tc);
	QueryPerformanceCounter(&t1);//��ִ��ʱ��start
	int j,i = 0;
	int sum = 0;//��¼������������
	int* partial_cnt = new int[N*N*N];//��¼���ֽ�
	int* L = new int[N];
	for(i=0; i<N; i++)
	{		
		L[i] = 0;
	}
	for(j=0; j<N*N*N; j++)		
			partial_cnt[j] = 0;//��ʼ��
	int* dev_cnt;
	int* dev_L;
	hipMalloc((void**)&dev_cnt,N*N*N*sizeof(int)); 
	hipMalloc((void**)&dev_L,N*sizeof(int)); 
	hipMemcpy(dev_cnt, partial_cnt,
                N*N*N*sizeof(int),
                hipMemcpyHostToDevice);

	HANDLE_ERROR( hipMemcpy(dev_L, L,
						N*sizeof(int),
						hipMemcpyHostToDevice) );

	QueryPerformanceFrequency(&tq);//
	QueryPerformanceCounter(&t3);//�˺�����ʼʱ��
	dim3 num_threads(N,N,N);
	kernel<<<1,num_threads>>>(dev_cnt,  dev_L);//ִ�к˺���
	QueryPerformanceCounter(&t4);//�˺�������ʱ��

	hipMemcpy(partial_cnt, dev_cnt, N*N*N*sizeof(int),hipMemcpyDeviceToHost);
	for(int k=0; k<N*N*N; k++)
	{
		//printf("Part = %d\n",partial_cnt[k]);
		sum += partial_cnt[k];	
	}
	printf("\n�ܹ�%d����\n", sum);

	QueryPerformanceCounter(&t2);//��ִ��ʱ��end
	printf("Use Time:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//��ӡ��ʱ
	printf("Kernel Use Time:%f\n",(t4.QuadPart-t3.QuadPart)*1.0/tq.QuadPart);//��ӡ�˺�����ʱ
	delete[]partial_cnt;
	delete[]L;
	hipFree(dev_cnt);
	hipFree(dev_L);
	return 0;
}

#endif