
#include <hip/hip_runtime.h>
#if 0
#include "E:\VisualStudio\CUDA\common\book.h"
#include <windows.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10

#define MAXSIZE 100 
//合法性判断——该row某列与其之前所有行比较，Q[]存储之前行合法列位置
__device__ int isLegal(int* L, int row, int pos )
{
	for(int i=0; i<row; i++)
	{
		if(L[i] == -1)return 0;
		if(pos == L[i])return 0;
		if(L[i] - pos == i - row)return 0;//正对角线
		if(L[i] - pos == row - i)return 0;//斜对角线
	}
	
	return 1;
}
//__device__ static int myCount = 0;//统计解个数
__device__ void findQueen(int* L, int row, int* count)
{
	int i;
	if(row == N)
	{
		return;
	}
	for(i=0; i<N; i++)
	{
		if(isLegal(L, row, i))
		{
			L[row] = i;//该行找到合法位置
			if(row == N-1)//递归到最后一行
			{
				(*count)++;
				//myCount++;
				L[row] = -1;//继续查找该行i列以后的位置
				//copyQueen
			}
			findQueen(L, row+1, count);//递归下一行
		}
	}
}
__device__ void Allocate(int* pa, int* pb, int* pc, int* count)
{
	int L[N] = {0};
	L[0] = *pa;
	L[1] = *pb;
	L[2] = *pc;
	findQueen(L, 3, count);
	
}

__global__ void kernel(int* part_sum, int* L)//*sum每个元素为部分解个数
{
	int z = threadIdx.z +  blockIdx.z * blockDim.z;
	int y = threadIdx.y +  blockIdx.y * blockDim.y;//块索引号
	int x = threadIdx.x +  blockIdx.x * blockDim.x;//线程索引号
	
	int a = y;
	int b = x;
	int c = z;
	
	//__shared__ int cache[N];//共享内存
	int tid = x + y*blockDim.x + z*blockDim.x*blockDim.x;
	//printf("(%d,%d,%d)\t",x,y,z);
	//printf("a,b,c = (%d, %d, %d)\n",a,b,c);
	int temp = 0;
	if(tid < N*N*N)
	{		
		//if(a != b && a-b != 1 && b-a != 1)
		{	
			if((a != b && a-b != 1 && b-a != 1)&&(c != b && c-b != 1 && b-c != 1)&&(a != c && a-c != 2 && c-a != 2))
				Allocate(&a, &b, &c, &temp);
		}
		//printf("Temp = %d\n",temp);
		
	}
	part_sum[tid] = temp;
	//printf("PartSUM = %d\n",part_sum[tid]);
	//printf("Tid = %d\t",tid);
	//__syncthreads();
	//归约运算
	// i = (N)/2;
	//while(i != 0)
	//{
	//	if(x <= i)
	//		cache[x] += cache[x + i];
	//	__syncthreads();
	//	i /= 2;
	//}	
}

int main()
{
	int j,i = 0;
	int sum = 0;//记录所有问题解个数
	int* partial_cnt = new int[N*N*N];//记录部分解
	int* L = new int[N];
	for(i=0; i<N; i++)
	{		
		L[i] = 0;
	}
	for(j=0; j<N*N*N; j++)		
			partial_cnt[j] = 0;//初始化
	int* dev_cnt;
	int* dev_L;
	cudaMalloc((void**)&dev_cnt,N*N*N*sizeof(int)); 
	cudaMalloc((void**)&dev_L,N*sizeof(int)); 
	cudaMemcpy(dev_cnt, partial_cnt,
                N*N*N*sizeof(int),
                cudaMemcpyHostToDevice);

	HANDLE_ERROR( cudaMemcpy(dev_L, L,
						N*sizeof(int),
						cudaMemcpyHostToDevice) );
	//dim3 num_threads(N,N,N);
	//kernel<<<1,num_threads>>>(dev_cnt,  dev_L);//执行核函数
	dim3 num_blocks((N+7)/8);
	dim3 num_threads(8,8,8);
	kernel<<<num_blocks,num_threads>>>(dev_cnt,  dev_L);//执行核函数
	cudaMemcpy(partial_cnt, dev_cnt, N*N*N*sizeof(int),cudaMemcpyDeviceToHost);
	for(int k=0; k<N*N*N; k++)
	{
		//printf("Part = %d\n",partial_cnt[k]);
		sum += partial_cnt[k];	
	}
	printf("\n总共%d个解\n", sum);
	delete[]partial_cnt;
	delete[]L;
	cudaFree(dev_cnt);
	cudaFree(dev_L);
	return 0;
}

#endif