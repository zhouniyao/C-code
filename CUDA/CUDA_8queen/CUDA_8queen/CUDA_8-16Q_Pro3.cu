
#include <hip/hip_runtime.h>
#if 0
#include "E:\VisualStudio\CUDA\common\book.h"
#include <windows.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10

#define MAXSIZE 100 
//�Ϸ����жϡ�����rowĳ������֮ǰ�����бȽϣ�Q[]�洢֮ǰ�кϷ���λ��
__device__ int isLegal(int* L, int row, int pos )
{
	for(int i=0; i<row; i++)
	{
		if(L[i] == -1)return 0;
		if(pos == L[i])return 0;
		if(L[i] - pos == i - row)return 0;//���Խ���
		if(L[i] - pos == row - i)return 0;//б�Խ���
	}
	
	return 1;
}
//__device__ static int myCount = 0;//ͳ�ƽ����
__device__ void findQueen(int* L, int row, int* count)
{
	int i;
	if(row == N)
	{
		return;
	}
	for(i=0; i<N; i++)
	{
		if(isLegal(L, row, i))
		{
			L[row] = i;//�����ҵ��Ϸ�λ��
			if(row == N-1)//�ݹ鵽���һ��
			{
				(*count)++;
				//myCount++;
				L[row] = -1;//�������Ҹ���i���Ժ��λ��
				//copyQueen
			}
			findQueen(L, row+1, count);//�ݹ���һ��
		}
	}
}
__device__ void Allocate(int* pa, int* pb, int* pc, int* count)
{
	int L[N] = {0};
	L[0] = *pa;
	L[1] = *pb;
	L[2] = *pc;
	findQueen(L, 3, count);
	
}

__global__ void kernel(int* part_sum, int* L)//*sumÿ��Ԫ��Ϊ���ֽ����
{
	int z = threadIdx.z +  blockIdx.z * blockDim.z;
	int y = threadIdx.y +  blockIdx.y * blockDim.y;//��������
	int x = threadIdx.x +  blockIdx.x * blockDim.x;//�߳�������
	
	int a = y;
	int b = x;
	int c = z;
	
	//__shared__ int cache[N];//�����ڴ�
	int tid = x + y*blockDim.x + z*blockDim.x*blockDim.x;
	//printf("(%d,%d,%d)\t",x,y,z);
	//printf("a,b,c = (%d, %d, %d)\n",a,b,c);
	int temp = 0;
	if(tid < N*N*N)
	{		
		//if(a != b && a-b != 1 && b-a != 1)
		{	
			if((a != b && a-b != 1 && b-a != 1)&&(c != b && c-b != 1 && b-c != 1)&&(a != c && a-c != 2 && c-a != 2))
				Allocate(&a, &b, &c, &temp);
		}
		//printf("Temp = %d\n",temp);
		
	}
	part_sum[tid] = temp;
	//printf("PartSUM = %d\n",part_sum[tid]);
	//printf("Tid = %d\t",tid);
	//__syncthreads();
	//��Լ����
	// i = (N)/2;
	//while(i != 0)
	//{
	//	if(x <= i)
	//		cache[x] += cache[x + i];
	//	__syncthreads();
	//	i /= 2;
	//}	
}

int main()
{
	int j,i = 0;
	int sum = 0;//��¼������������
	int* partial_cnt = new int[N*N*N];//��¼���ֽ�
	int* L = new int[N];
	for(i=0; i<N; i++)
	{		
		L[i] = 0;
	}
	for(j=0; j<N*N*N; j++)		
			partial_cnt[j] = 0;//��ʼ��
	int* dev_cnt;
	int* dev_L;
	cudaMalloc((void**)&dev_cnt,N*N*N*sizeof(int)); 
	cudaMalloc((void**)&dev_L,N*sizeof(int)); 
	cudaMemcpy(dev_cnt, partial_cnt,
                N*N*N*sizeof(int),
                cudaMemcpyHostToDevice);

	HANDLE_ERROR( cudaMemcpy(dev_L, L,
						N*sizeof(int),
						cudaMemcpyHostToDevice) );
	//dim3 num_threads(N,N,N);
	//kernel<<<1,num_threads>>>(dev_cnt,  dev_L);//ִ�к˺���
	dim3 num_blocks((N+7)/8);
	dim3 num_threads(8,8,8);
	kernel<<<num_blocks,num_threads>>>(dev_cnt,  dev_L);//ִ�к˺���
	cudaMemcpy(partial_cnt, dev_cnt, N*N*N*sizeof(int),cudaMemcpyDeviceToHost);
	for(int k=0; k<N*N*N; k++)
	{
		//printf("Part = %d\n",partial_cnt[k]);
		sum += partial_cnt[k];	
	}
	printf("\n�ܹ�%d����\n", sum);
	delete[]partial_cnt;
	delete[]L;
	cudaFree(dev_cnt);
	cudaFree(dev_L);
	return 0;
}

#endif