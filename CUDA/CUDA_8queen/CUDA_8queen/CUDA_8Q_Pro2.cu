
#include <hip/hip_runtime.h>
#if 0
/*
*N�ʺ�����CUDA����ͳ�Ĵ����㷨�����ݹ���ݣ����л����滮���2��
*
*
*/
#include "E:\VisualStudio\CUDA\common\book.h"
#include <windows.h>
#include <stdio.h>
#include <stdlib.h>

#define N 8
#define MAXSIZE 100 
//�Ϸ����жϡ�����rowĳ������֮ǰ�����бȽϣ�Q[]�洢֮ǰ�кϷ���λ��
__device__ int isLegal(int* L, int row, int pos )
{
	for(int i=0; i<row; i++)
	{
		if(L[i] == -1)return 0;
		if(pos == L[i])return 0;
		if(L[i] - pos == i - row)return 0;//���Խ���
		if(L[i] - pos == row - i)return 0;//б�Խ���
	}
	
	return 1;
}
//__device__ static int myCount = 0;//ͳ�ƽ����
__device__ void findQueen(int* L, int row, int* count)
{
	int i;
	if(row == N)
	{
		return;
	}
	for(i=0; i<N; i++)
	{
		if(isLegal(L, row, i))
		{
			L[row] = i;//�����ҵ��Ϸ�λ��
			if(row == N-1)//�ݹ鵽���һ��
			{
				(*count)++;
				//myCount++;
				L[row] = -1;//�������Ҹ���i���Ժ��λ��
				//copyQueen
			}
			findQueen(L, row+1, count);//�ݹ���һ��
		}
	}
}
__device__ void Allocate(int* pa, int* pb, int* count)
{
	int L[N] = {0};
	L[0] = *pa;
	L[1] = *pb;
	findQueen(L, 2, count);
	
}

__global__ void kernel(int* part_sum, int* L)//*sumÿ��Ԫ��Ϊ���ֽ����
{
	int y = blockIdx.x;//��������
	int x = threadIdx.x;//�߳�������
	
	int a = y;
	int b = x;
	//__shared__ int cache[N];//�����ڴ�
	int tid = x+y*blockDim.x;
	int temp = 0;
	//if(tid < N*N)
	{		
		if(a != b && a-b != 1 && b-a != 1)
		{			
			Allocate(&a, &b, &temp);
		}
		//printf("Temp = %d\n",temp);
		part_sum[tid] = temp;
	}
	//printf("PartSUM = %d\n",part_sum[tid]);
	//__syncthreads();
	//��Լ����
	// i = (N)/2;
	//while(i != 0)
	//{
	//	if(x <= i)
	//		cache[x] += cache[x + i];
	//	__syncthreads();
	//	i /= 2;
	//}	
}

int main()
{
	LARGE_INTEGER t1,t2,tc,t3,t4,tq;//����ʱ�����
	QueryPerformanceFrequency(&tc);
	QueryPerformanceCounter(&t1);//��ִ��ʱ��start
	int j,i = 0;
	int sum = 0;//��¼������������
	int* partial_cnt = new int[N*N];//��¼���ֽ�
	int* L = new int[N];
	for(i=0; i<N; i++)
	{		
		L[i] = 0;
	}
	for(j=0; j<N*N; j++)		
			partial_cnt[j] = 0;//��ʼ��
	int* dev_cnt;
	int* dev_L;
	cudaMalloc((void**)&dev_cnt,N*N*sizeof(int)); 
	cudaMalloc((void**)&dev_L,N*sizeof(int)); 
	cudaMemcpy(dev_cnt, partial_cnt,
                N*sizeof(int),
                cudaMemcpyHostToDevice);


	//for(i=0; i<N; i++)
	//{
	//	L[0] = i;
	//	for(j=0; j<N; j++)
	//	{
	//		if(j != L[0] && L[0]-j != 1 && j-L[0] != 1)
	//		{
	//			L[1] = j;
	//			HANDLE_ERROR( cudaMemcpy(dev_L, L,
	//								N*sizeof(int),
	//								cudaMemcpyHostToDevice) );
	//			kernel<<<1,1>>>(dev_cnt,  dev_L);//ִ�к˺���
	//			cudaMemcpy(partial_cnt, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
	//			for(int k=0; k<N; k++)
	//				sum += partial_cnt[k];	
	//		}
	//	}
	//}

	HANDLE_ERROR( cudaMemcpy(dev_L, L,
						N*sizeof(int),
						cudaMemcpyHostToDevice) );
	QueryPerformanceFrequency(&tq);//
	QueryPerformanceCounter(&t3);//�˺�����ʼʱ��
	kernel<<<N,N>>>(dev_cnt,  dev_L);//ִ�к˺���
	QueryPerformanceCounter(&t4);//�˺�������ʱ��

	cudaMemcpy(partial_cnt, dev_cnt, N*N*sizeof(int),cudaMemcpyDeviceToHost);
	for(int k=0; k<N*N; k++)
	{
		//printf("Part = %d\n",partial_cnt[k]);
		sum += partial_cnt[k];	
	}
	printf("\n�ܹ�%d����\n", sum);
	QueryPerformanceCounter(&t2);//��ִ��ʱ��end
	printf("Use Time:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//��ӡ��ʱ
	printf("Kernel Use Time:%f\n",(t4.QuadPart-t3.QuadPart)*1.0/tq.QuadPart);//��ӡ�˺�����ʱ
	return 0;
}

//�״θĽ���������
//int main()
//{
//	int i=0,sum=0;
//	int count[N] = {0};
//	int* L = new int[N];
//	for(i=0; i<N; i++)
//		L[i] = 0;
//	int* dev_cnt;
//	int* dev_L;
//	cudaMalloc((void**)&dev_cnt,N*sizeof(int)); 
//	cudaMalloc((void**)&dev_L,N*sizeof(int)); 
//	cudaMemcpy(dev_cnt, count,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//
//	//HANDLE_ERROR( cudaMemcpy(dev_L, L,
// //                             N*sizeof(int),
// //                             cudaMemcpyHostToDevice) );
//			/*ʱ���ʱ����*/
//	LARGE_INTEGER t1,t2,tc;//����ʱ�����
//	QueryPerformanceFrequency(&tc);//
//	QueryPerformanceCounter(&t1);//
//	i = 0;
//
//	switch(N)
//	{
////===================4�ʺ�========================
//	case 4:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//
////=================5�ʺ�==========================
//		case 5:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================6�ʺ�==========================
//		case 6:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================7�ʺ�==========================
//		case 7:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//	break;
//	//=================8�ʺ�==========================
//		case 8:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================9�ʺ�==========================
//		case 9:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//	break;
//	//=================10�ʺ�==========================
//		case 10:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================11�ʺ�==========================
//		case 11:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 10;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//	break;
//	//=================12�ʺ�==========================
//		case 12:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 10;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//								L[0] = 11;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================13�ʺ�==========================
//		case 13:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 10;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 11;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 12;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================14�ʺ�==========================
//		case 14:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 10;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 11;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 12;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 13;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================15�ʺ�==========================
//		case 15:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 10;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 11;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 12;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 13;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 14;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================16�ʺ�==========================
//		case 16:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 10;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//		
//								L[0] = 11;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 12;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 13;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 14;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//										L[0] = 15;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//
//	}
//	QueryPerformanceCounter(&t2);//����ʱ��
//	printf("\n�ܹ�%d����\n", sum);
//
//	printf("Use Time:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//��ӡ��ʱ
//	cudaFree(dev_cnt);
//	delete []L;
//	return 0;
//}
#endif