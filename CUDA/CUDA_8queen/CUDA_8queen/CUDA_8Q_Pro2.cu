
#include <hip/hip_runtime.h>
#if 0
/*
*N皇后，利用CUDA将传统的串行算法——递归回溯，并行化，规划最初2层
*
*
*/
#include "E:\VisualStudio\CUDA\common\book.h"
#include <windows.h>
#include <stdio.h>
#include <stdlib.h>

#define N 8
#define MAXSIZE 100 
//合法性判断——该row某列与其之前所有行比较，Q[]存储之前行合法列位置
__device__ int isLegal(int* L, int row, int pos )
{
	for(int i=0; i<row; i++)
	{
		if(L[i] == -1)return 0;
		if(pos == L[i])return 0;
		if(L[i] - pos == i - row)return 0;//正对角线
		if(L[i] - pos == row - i)return 0;//斜对角线
	}
	
	return 1;
}
//__device__ static int myCount = 0;//统计解个数
__device__ void findQueen(int* L, int row, int* count)
{
	int i;
	if(row == N)
	{
		return;
	}
	for(i=0; i<N; i++)
	{
		if(isLegal(L, row, i))
		{
			L[row] = i;//该行找到合法位置
			if(row == N-1)//递归到最后一行
			{
				(*count)++;
				//myCount++;
				L[row] = -1;//继续查找该行i列以后的位置
				//copyQueen
			}
			findQueen(L, row+1, count);//递归下一行
		}
	}
}
__device__ void Allocate(int* pa, int* pb, int* count)
{
	int L[N] = {0};
	L[0] = *pa;
	L[1] = *pb;
	findQueen(L, 2, count);
	
}

__global__ void kernel(int* part_sum, int* L)//*sum每个元素为部分解个数
{
	int y = blockIdx.x;//块索引号
	int x = threadIdx.x;//线程索引号
	
	int a = y;
	int b = x;
	//__shared__ int cache[N];//共享内存
	int tid = x+y*blockDim.x;
	int temp = 0;
	//if(tid < N*N)
	{		
		if(a != b && a-b != 1 && b-a != 1)
		{			
			Allocate(&a, &b, &temp);
		}
		//printf("Temp = %d\n",temp);
		part_sum[tid] = temp;
	}
	//printf("PartSUM = %d\n",part_sum[tid]);
	//__syncthreads();
	//归约运算
	// i = (N)/2;
	//while(i != 0)
	//{
	//	if(x <= i)
	//		cache[x] += cache[x + i];
	//	__syncthreads();
	//	i /= 2;
	//}	
}

int main()
{
	LARGE_INTEGER t1,t2,tc,t3,t4,tq;//定义时间变量
	QueryPerformanceFrequency(&tc);
	QueryPerformanceCounter(&t1);//总执行时间start
	int j,i = 0;
	int sum = 0;//记录所有问题解个数
	int* partial_cnt = new int[N*N];//记录部分解
	int* L = new int[N];
	for(i=0; i<N; i++)
	{		
		L[i] = 0;
	}
	for(j=0; j<N*N; j++)		
			partial_cnt[j] = 0;//初始化
	int* dev_cnt;
	int* dev_L;
	cudaMalloc((void**)&dev_cnt,N*N*sizeof(int)); 
	cudaMalloc((void**)&dev_L,N*sizeof(int)); 
	cudaMemcpy(dev_cnt, partial_cnt,
                N*sizeof(int),
                cudaMemcpyHostToDevice);


	//for(i=0; i<N; i++)
	//{
	//	L[0] = i;
	//	for(j=0; j<N; j++)
	//	{
	//		if(j != L[0] && L[0]-j != 1 && j-L[0] != 1)
	//		{
	//			L[1] = j;
	//			HANDLE_ERROR( cudaMemcpy(dev_L, L,
	//								N*sizeof(int),
	//								cudaMemcpyHostToDevice) );
	//			kernel<<<1,1>>>(dev_cnt,  dev_L);//执行核函数
	//			cudaMemcpy(partial_cnt, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
	//			for(int k=0; k<N; k++)
	//				sum += partial_cnt[k];	
	//		}
	//	}
	//}

	HANDLE_ERROR( cudaMemcpy(dev_L, L,
						N*sizeof(int),
						cudaMemcpyHostToDevice) );
	QueryPerformanceFrequency(&tq);//
	QueryPerformanceCounter(&t3);//核函数开始时间
	kernel<<<N,N>>>(dev_cnt,  dev_L);//执行核函数
	QueryPerformanceCounter(&t4);//核函数结束时间

	cudaMemcpy(partial_cnt, dev_cnt, N*N*sizeof(int),cudaMemcpyDeviceToHost);
	for(int k=0; k<N*N; k++)
	{
		//printf("Part = %d\n",partial_cnt[k]);
		sum += partial_cnt[k];	
	}
	printf("\n总共%d个解\n", sum);
	QueryPerformanceCounter(&t2);//总执行时间end
	printf("Use Time:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//打印耗时
	printf("Kernel Use Time:%f\n",(t4.QuadPart-t3.QuadPart)*1.0/tq.QuadPart);//打印核函数耗时
	return 0;
}

//首次改进——有误
//int main()
//{
//	int i=0,sum=0;
//	int count[N] = {0};
//	int* L = new int[N];
//	for(i=0; i<N; i++)
//		L[i] = 0;
//	int* dev_cnt;
//	int* dev_L;
//	cudaMalloc((void**)&dev_cnt,N*sizeof(int)); 
//	cudaMalloc((void**)&dev_L,N*sizeof(int)); 
//	cudaMemcpy(dev_cnt, count,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//
//	//HANDLE_ERROR( cudaMemcpy(dev_L, L,
// //                             N*sizeof(int),
// //                             cudaMemcpyHostToDevice) );
//			/*时间计时函数*/
//	LARGE_INTEGER t1,t2,tc;//定义时间变量
//	QueryPerformanceFrequency(&tc);//
//	QueryPerformanceCounter(&t1);//
//	i = 0;
//
//	switch(N)
//	{
////===================4皇后========================
//	case 4:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//
////=================5皇后==========================
//		case 5:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================6皇后==========================
//		case 6:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================7皇后==========================
//		case 7:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//	break;
//	//=================8皇后==========================
//		case 8:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================9皇后==========================
//		case 9:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//	break;
//	//=================10皇后==========================
//		case 10:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================11皇后==========================
//		case 11:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 10;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//	break;
//	//=================12皇后==========================
//		case 12:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 10;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//								L[0] = 11;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================13皇后==========================
//		case 13:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 10;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 11;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 12;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================14皇后==========================
//		case 14:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 10;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 11;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 12;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 13;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================15皇后==========================
//		case 15:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 10;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 11;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 12;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 13;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 14;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//	//=================16皇后==========================
//		case 16:
//		L[0] = 0;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 1;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//		
//				L[0] = 2;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//				L[0] = 3;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 4;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 5;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 6;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 7;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 8;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 9;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//						L[0] = 10;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//		
//								L[0] = 11;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 12;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 13;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//								L[0] = 14;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//										L[0] = 15;
//		cudaMemcpy(dev_L, L,
//                N*sizeof(int),
//                cudaMemcpyHostToDevice);
//		kernel<<<1,N>>>(dev_cnt, dev_L);
//		cudaMemcpy(count, dev_cnt, N*sizeof(int),cudaMemcpyDeviceToHost);
//		for(i=0; i<N; i++)
//			sum += count[i];
//
//	break;
//
//	}
//	QueryPerformanceCounter(&t2);//结束时间
//	printf("\n总共%d个解\n", sum);
//
//	printf("Use Time:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//打印耗时
//	cudaFree(dev_cnt);
//	delete []L;
//	return 0;
//}
#endif