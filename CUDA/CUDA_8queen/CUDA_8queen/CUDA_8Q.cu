
#include <hip/hip_runtime.h>
#if 0


/*
*N�ʺ�����CUDA����ͳ�Ĵ����㷨����äĿ��٣����л�
*
*
*/
#include "E:\VisualStudio\CUDA\common\book.h"
#include <windows.h>
#include <stdio.h>
#include <stdlib.h>

#define N 8

#define MAXSIZE 100
//int (*Queen)[N] = new int[INT_MAX/100][N];
//int (*Queen)[N] = (int (*)[N])malloc(MAXSIZE*N*sizeof(int));

//��ʼ����ά����
__device__ void InitMatrix(int L[][N])
{
	int i,j;
	for(i=0; i<N; i++)
		for(j=0; j<N; j++)
			L[i][j] = 0;
}
//�����ס������ϵĸ�
__device__ void Mi(int L[][N], int row, int col)
{
	int i,j;
	for(i=0; i<N; i++)
	{
		L[row][i] = -1;//ͬ��
		L[i][col] = -1;//ͬ��
	}
	//��б��	//��б��
	for(i=0; i<N; i++)
		for(j=0; j<N; j++)
			if((i-row == col-j)||(i-row == j-col))
				L[i][j] = -1;
}
//�Ϸ����жϡ�����rowĳ������֮ǰ�����бȽϣ�Q[]�洢֮ǰ�кϷ���λ��
__device__ int isLegal(int L[], int row, int pos )
{
	for(int i=0; i<row; i++)
	{
		if(L[i] == -1)return 0;
		if(pos == L[i])return 0;
		if(L[i] - pos == i - row)return 0;//���Խ���
		if(L[i] - pos == row - i)return 0;//б�Խ���
	}
	
	return 1;
}
//int isLegal(int Q[], int row, int col)
//{
//	int i;
//	if(row == 0)return 1;
//	for(i=0; i<row; i++)
//		if(col==Q[i] || col-i == row-Q[i] || i-col == row-Q[i])
//			return 0;
//	return 1;
//	
//}
//��������_������н�����ά����ĳ��
//__device__ void saveQueen(int Queen[][N], int cnt, int L[])
//{	
//	for(int i=0; i<N; i++)
//		Queen[cnt][i] = L[i];	
//}
__device__ static int myCount = 0;//ͳ�ƽ������û����

//�̶�һ�����Ӻ󣬲��Ҹ�״̬�µ����п��ܣ����ؽ����
/***************************************** 4�ʺ�*****************************************/
//__device__ int findQueen4(int row, int col)//rowΪ0��colΪthreadIdx
//{
//	//int i,j,flag;
//	//int L[N][N];
//	//int temp[N];
//	///*���̳�ʼ��*/
//	////��б��	//��б��
//	//for(i=0; i<N; i++)
//	//	for(j=0; j<N; j++)
//	//		if((i-row == col-j)||(i-row == j-col))
//	//			L[i][j] = -1;
//	//		else
//	//			L[i][j]=0;
//
//	//for(i=0; i<N; i++)
//	//{
//	//	L[row][i] = -1;//ͬ��
//	//	L[i][col] = -1;//ͬ��
//	//}
//	int cnt = 0;
//	int L[N] = {0};
//	L[row] = col;
//	for(int i1=0; i1<N; i1++)/*��1��*/
//	{
//		//if(L[1][i1]==-1)continue;		
//		if(isLegal(L, 1, i1 ))
//		{
//			L[1] = i1;
//			for(int i2=0; i2<N; i2++)/*��2��*/
//			{
//				
//				//if(L[2][i2]==-1)continue;
//				
//				if(isLegal(L, 2, i2 ))
//				{
//					L[2] = i2;
//					for(int i3=0; i3<N; i3++)/*��3��*/
//					{
//						
//						//if(L[3][i3]==-1)continue;
//						if(isLegal(L, 3, i3 ))
//						{
//							L[3] = i3;
//							cnt++;
//							myCount++;
//						}
//			
//					}
//				}
//			}
//		}
//	}
//				
//	return cnt;
//}
__device__ int findQueen4(int row, int col)
{

	int  i,i0, i1, i2, i3;
	int cnt = 0;
	int Q[N] = {0};
	int L[N][N] = {0};
	//for(i=0; i<N; i++)
	//	Q[i] = 0;
	//InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(row==3)goto Layer4;
												else
													if(L[3][i3] != -1 && isLegal(Q, 3, i3))
													{
														Q[3] = i3;
		Layer4:											//����Ϊ���н�
														cnt++;
														//myCount++;
														//saveQueen(Queen,myCount,Q);
													}
										}
							}
			}	
	return cnt;
}
//***************************************** 5�ʺ�*****************************************/
__device__ int findQueen5(int row, int col)
{
	int L[N][N] = {0};
	int i,i0, i1, i2, i3, i4;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
																//����Ϊ���н�
																cnt++;
																//myCount++;
																//saveQueen(Queen,myCount,Q);
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 6�ʺ�*****************************************/
__device__ int findQueen6(int row, int col)
{
	int L[N][N] = {0};
	int i,i0, i1, i2, i3, i4, i5;
	int Q[N] = {0};
	int cnt = 0;

	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																			//����Ϊ���н�
																			cnt++;
																			//myCount++;
																			//saveQueen(Queen,myCount,Q);
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 7�ʺ�*****************************************/
__device__ int findQueen7(int row, int col )
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																					//����Ϊ���н�
																					cnt++;
																					//myCount++;
																					//saveQueen(Queen,myCount,Q);
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 8�ʺ�*****************************************/
__device__ int findQueen8(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																							//����Ϊ���н�
																							cnt++;
																							//myCount++;
																							//saveQueen(Queen,myCount,Q);
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
///***************************************** 9�ʺ�*****************************************/
__device__ int findQueen9(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8;
	int Q[N] = {0} ;
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*��8��*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								//����Ϊ���н�
																								cnt++;
																								//myCount++;
																								//saveQueen(Queen,myCount,Q);
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt ;
}
//***************************************** 10�ʺ�*****************************************/
__device__ int findQueen10(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*��8��*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*��9��*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										//����Ϊ���н�
																										cnt++;
																										//myCount++;
																										//saveQueen(Queen,myCount,Q);
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 11�ʺ�*****************************************/
__device__ int findQueen11(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*��8��*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*��9��*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*��10��*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												//����Ϊ���н�
																												cnt++;
																												//myCount++;
																												//saveQueen(Queen,myCount,Q);
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 12�ʺ�*****************************************/
__device__ int findQueen12(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*��8��*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*��9��*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*��10��*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*��11��*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														//����Ϊ���н�
																														cnt++;
																														//myCount++;
																														//saveQueen(Queen,myCount,Q);
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 13�ʺ�*****************************************/
__device__ int findQueen13(int row, int col )
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*��8��*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*��9��*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*��10��*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*��11��*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*��12��*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																//����Ϊ���н�
																																cnt++;
																																//myCount++;
																																//saveQueen(Queen,myCount,Q);
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 14�ʺ�*****************************************/
__device__ int findQueen14(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*��8��*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*��9��*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*��10��*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*��11��*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*��12��*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*��13��*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		//����Ϊ���н�
																																		cnt++;
																																		//myCount++;
																																		//saveQueen(Queen,myCount,Q);
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 15�ʺ�*****************************************/
//int findQueen15(int row, int col)
//{
//	int L[N][N], i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14;
//	int Q[N],count=-1;
//	for(i=0; i<N; i++)
//		Q[i] = 0;
//	InitMatrix(L);//��ʼ������
//	Mi(L, row, col);//�����ס������ϵĸ�
//	Q[row] = col;//��¼ָ������
//	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�
//
//		//for(i0=0; i0<N; i0++)/*��0��*/
//			if(row==0)goto Layer1;
//			else
//				if(L[0][i0] != -1)//���ӿɷ�λ��
//				{
//					Q[0] = i0;
//
//		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
//						if(row==1)goto Layer2;
//						else
//							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
//							{
//								Q[1] = i1;
//
//		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
//									if(row==2)goto Layer3;
//									else
//										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
//										{
//											Q[2] = i2;
//
//		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
//												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
//												{
//													Q[3] = i3;
//													for(i4=0; i4<N; i4++)/*��4��*/
//													{
//														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
//														{
//															Q[4] = i4;
//															for(i5=0; i5<N; i5++)/*��5��*/
//															{																
//																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
//																	{
//																		Q[5] = i5;
//																		for(i6=0; i6<N; i6++)/*��6��*/
//																		{
//																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
//																			{
//																				Q[6] = i6;
//																				for(i7=0; i7<N; i7++)/*��7��*/
//																				{
//																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
//																					{
//																						Q[7] = i7;
//																						for(i8=0; i8<N; i8++)/*��8��*/
//																						{
//																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
//																							{
//																								Q[8] = i8;
//																								for(i9=0; i9<N; i9++)/*��9��*/
//																								{
//																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
//																									{
//																										Q[9] = i9;
//																										for(i10=0; i10<N; i10++)/*��10��*/
//																										{
//																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
//																											{
//																												Q[10] = i10;
//																												for(i11=0; i11<N; i11++)/*��11��*/
//																												{
//																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
//																													{
//																														Q[11] = i11;
//																														for(i12=0; i12<N; i12++)/*��12��*/
//																														{
//																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
//																															{
//																																Q[12] = i12;
//																																for(i13=0; i13<N; i13++)/*��13��*/
//																																{
//																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
//																																	{
//																																		Q[13] = i13;
//																																		for(i14=0; i14<N; i14++)/*��14��*/
//																																		{
//																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
//																																			{
//																																				Q[14] = i14;
//																																				//����Ϊ���н�
//																																				count++;
//																																				myCount++;
//																																				saveQueen(Queen,myCount,Q);
//																																			}
//																																		}
//																																	}
//																																}
//																															}
//																														}
//																													}
//																												}
//																											}
//																										}
//																									}
//																								}
//																							}
//																						}
//																					}
//																				}
//																			}
//																		}
//																	}
//															}
//														}
//													}
//												}
//										}
//							}
//			}	
//	return count+1;
//}
__device__ int findQueen15(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*��8��*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*��9��*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*��10��*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*��11��*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*��12��*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*��13��*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		for(i14=0; i14<N; i14++)/*��14��*/
																																		{
																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
																																			{
																																				Q[14] = i14;
																																				//����Ϊ���н�
																																				cnt++;
																																				//myCount++;
																																				//saveQueen(Queen,myCount,Q);
																																			}
																																		}
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 16�ʺ�*****************************************/
__device__ int findQueen16(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14, i15;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*��8��*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*��9��*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*��10��*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*��11��*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*��12��*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*��13��*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		for(i14=0; i14<N; i14++)/*��14��*/
																																		{
																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
																																			{
																																				Q[14] = i14;
																																				for(i15=0; i15<N; i15++)/*��15��*/
																																				{
																																					if(L[15][i15] != -1 && isLegal(Q, 15, i15))
																																					{
																																						Q[15] = i15;
																																						//����Ϊ���н�
																																						cnt++;
																																						//myCount++;
																																						//saveQueen(Queen,myCount,Q);
																																					}
																																				}
																																			}
																																		}
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt ;
}
//***************************************** 17�ʺ�*****************************************/
__device__ int findQueen17(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14, i15, i16;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*��8��*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*��9��*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*��10��*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*��11��*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*��12��*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*��13��*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		for(i14=0; i14<N; i14++)/*��14��*/
																																		{
																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
																																			{
																																				Q[14] = i14;
																																				for(i15=0; i15<N; i15++)/*��15��*/
																																				{
																																					if(L[15][i15] != -1 && isLegal(Q, 15, i15))
																																					{
																																						Q[15] = i15;
																																						for(i16=0; i16<N; i16++)/*��16��*/
																																						{
																																							if(L[16][i16] != -1 && isLegal(Q, 16, i16))
																																							{
																																								Q[16] = i16;
																																								//����Ϊ���н�
																																								cnt++;
																																								//myCount++;
																																								//saveQueen(Queen,myCount,Q);
																																							}
																																						}
																																					}
																																				}
																																			}
																																		}
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 18�ʺ�*****************************************/
__device__ int findQueen18(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14, i15, i16, i17;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*��8��*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*��9��*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*��10��*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*��11��*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*��12��*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*��13��*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		for(i14=0; i14<N; i14++)/*��14��*/
																																		{
																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
																																			{
																																				Q[14] = i14;
																																				for(i15=0; i15<N; i15++)/*��15��*/
																																				{
																																					if(L[15][i15] != -1 && isLegal(Q, 15, i15))
																																					{
																																						Q[15] = i15;
																																						for(i16=0; i16<N; i16++)/*��16��*/
																																						{
																																							if(L[16][i16] != -1 && isLegal(Q, 16, i16))
																																							{
																																								Q[16] = i16;
																																								for(i17=0; i17<N; i17++)/*��17��*/
																																								{
																																									if(L[17][i17] != -1 && isLegal(Q, 17, i17))
																																									{
																																										Q[17] = i17;
																																										//����Ϊ���н�
																																										cnt++;
																																										//myCount++;
																																										//saveQueen(Queen,myCount,Q);
																																									}
																																								}
																																							}
																																						}
																																					}
																																				}
																																			}
																																		}
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt ;
}
//***************************************** 19�ʺ�*****************************************/
__device__ int findQueen19(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14, i15, i16, i17, i18;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*��8��*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*��9��*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*��10��*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*��11��*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*��12��*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*��13��*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		for(i14=0; i14<N; i14++)/*��14��*/
																																		{
																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
																																			{
																																				Q[14] = i14;
																																				for(i15=0; i15<N; i15++)/*��15��*/
																																				{
																																					if(L[15][i15] != -1 && isLegal(Q, 15, i15))
																																					{
																																						Q[15] = i15;
																																						for(i16=0; i16<N; i16++)/*��16��*/
																																						{
																																							if(L[16][i16] != -1 && isLegal(Q, 16, i16))
																																							{
																																								Q[16] = i16;
																																								for(i17=0; i17<N; i17++)/*��17��*/
																																								{
																																									if(L[17][i17] != -1 && isLegal(Q, 17, i17))
																																									{
																																										Q[17] = i17;
																																										for(i18=0; i18<N; i18++)/*��18��*/
																																										{
																																											if(L[18][i18] != -1 && isLegal(Q, 18, i18))
																																											{
																																												Q[18] = i18;
																																												//����Ϊ���н�
																																												cnt++;
																																												//myCount++;
																																												//saveQueen(Queen,myCount,Q);
																																											}
																																										}
																																									}
																																								}
																																							}
																																						}
																																					}
																																				}
																																			}
																																		}
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 20�ʺ�*****************************************/
__device__ int findQueen20(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14, i15, i16, i17, i18, i19;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//��ʼ������
	Mi(L, row, col);//�����ס������ϵĸ�
	Q[row] = col;//��¼ָ������
	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�

		//for(i0=0; i0<N; i0++)/*��0��*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//���ӿɷ�λ��
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*��1��*/      /*����ѭ����ȴѭ����4�Σ��õ�4����ͬ���*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*��4��*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*��5��*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*��6��*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*��7��*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*��8��*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*��9��*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*��10��*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*��11��*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*��12��*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*��13��*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		for(i14=0; i14<N; i14++)/*��14��*/
																																		{
																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
																																			{
																																				Q[14] = i14;
																																				for(i15=0; i15<N; i15++)/*��15��*/
																																				{
																																					if(L[15][i15] != -1 && isLegal(Q, 15, i15))
																																					{
																																						Q[15] = i15;
																																						for(i16=0; i16<N; i16++)/*��16��*/
																																						{
																																							if(L[16][i16] != -1 && isLegal(Q, 16, i16))
																																							{
																																								Q[16] = i16;
																																								for(i17=0; i17<N; i17++)/*��17��*/
																																								{
																																									if(L[17][i17] != -1 && isLegal(Q, 17, i17))
																																									{
																																										Q[17] = i17;
																																										for(i18=0; i18<N; i18++)/*��18��*/
																																										{
																																											if(L[18][i18] != -1 && isLegal(Q, 18, i18))
																																											{
																																												Q[18] = i18;
																																												for(i19=0; i19<N; i19++)/*��19��*/
																																												{
																																													if(L[19][i19] != -1 && isLegal(Q, 19, i19))
																																													{
																																														Q[19] = i19;
																																														//����Ϊ���н�
																																														cnt++;
																																														//myCount++;
																																														//saveQueen(Queen,myCount,Q);
																																													}
																																												}
																																											}
																																										}
																																									}
																																								}
																																							}
																																						}
																																					}
																																				}
																																			}
																																		}
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}

//��ӡ�ʺ�
void showQueen(int Queen[][N], int count)
{
	for(int i=0; i<=count; i++)
	{
		printf("��%d���⣺ ",i+1);
		for(int j=0; j<N; j++)
			printf("(%d,%d)",j,Queen[i][j]);
		printf("\n");
	}
}

//�������
__global__ void kernel4(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen4(0, i);
			
}
__global__ void kernel5(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen5(0, i);
			
}
__global__ void kernel6(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen6(0, i);
			
}
__global__ void kernel7(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen7(0, i);
			
}
__global__ void kernel8(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen8(0, i);
			
}
__global__ void kernel9(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen9(0, i);
			
}
__global__ void kernel10(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen10(0, i);
			
}
__global__ void kernel11(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen11(0, i);
			
}
__global__ void kernel12(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen12(0, i);
			
}
__global__ void kernel13(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen13(0, i);
			
}
__global__ void kernel14(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen14(0, i);
			
}
__global__ void kernel15(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen15(0, i);
			
}
__global__ void kernel16(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen16(0, i);
			
}
__global__ void kernel17(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen17(0, i);
			
}
__global__ void kernel18(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen18(0, i);
			
}
__global__ void kernel19(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen19(0, i);
			
}
__global__ void kernel20(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen20(0, i);
			
}
int main()
{
	LARGE_INTEGER t1,t2,tc,t3,t4;//����ʱ�����
	QueryPerformanceFrequency(&tc);
	QueryPerformanceCounter(&t1);//��ִ��ʱ��start
	int i=0,sum=0;
	//int* dev_row;
	//int* dev_col;
	//int count[N] = {0};
	int* count = (int*)malloc(N*sizeof(int));
	for(i=0; i<N; i++)
		*(count+i) = 0;
	int* dev_c;
	//HANDLE_ERROR(cudaMalloc((void**)&dev_row,sizeof(int)));//��
	//HANDLE_ERROR(cudaMalloc((void**)&dev_col,sizeof(int)));//��
	HANDLE_ERROR(cudaMalloc((void**)&dev_c,N*sizeof(int)));
    HANDLE_ERROR( cudaMemcpy(dev_c, count,
                              N*sizeof(int),
                              cudaMemcpyHostToDevice) );

	QueryPerformanceCounter(&t2);//�˺�����ʼʱ��
	switch(N)
	{
	case 4:
		//Allocate<<<1,N>>>(dev_c);
		kernel4<<<N,1>>>(dev_c);
		break;
	case 5:
		kernel5<<<N,1>>>(dev_c);
		break;
	case 6:
		kernel6<<<N,1>>>(dev_c);
		break;
	case 7:
		kernel7<<<N,1>>>(dev_c);
		break;
	case 8:
		kernel8<<<N,1>>>(dev_c);
		break;
	case 9:
		kernel9<<<N,1>>>(dev_c);
		break;
	case 10:
		kernel10<<<N,1>>>(dev_c);
		break;
	case 11:
		kernel11<<<N,1>>>(dev_c);
		break;
	case 12:
		kernel12<<<N,1>>>(dev_c);
		break;
	case 13:
		kernel13<<<N,1>>>(dev_c);
		break;
	case 14:
		kernel14<<<N,1>>>(dev_c);
	case 15:
		kernel15<<<N,1>>>(dev_c);
	case 16:
		kernel16<<<N,1>>>(dev_c);
	case 17:
		kernel17<<<N,1>>>(dev_c);
		break;
	case 18:
		kernel18<<<N,1>>>(dev_c);
		break;
	case 19:
		kernel19<<<N,1>>>(dev_c);
		break;
	case 20:
		kernel20<<<N,1>>>(dev_c);
			

		default:
			;
	
	}
	QueryPerformanceCounter(&t3);//�˺�������ʱ��
	HANDLE_ERROR(cudaMemcpy(count,dev_c,N*sizeof(int),cudaMemcpyDeviceToHost));

	for(i=0; i<N; i++ )
		sum += count[i];
	//printf("\n�ܹ�%d����\n", sum2);
	printf("\n�ܹ�%d����\n", sum);
	QueryPerformanceCounter(&t4);//����ʱ��
	printf("Use Time:%f\n",(t4.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//��ӡ��ʱ
	printf("Kernel Use Time:%f\n",(t3.QuadPart-t2.QuadPart)*1.0/tc.QuadPart);//��ӡ�˺�����ʱ

	//printf("XX1:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//��ӡ�˺�����ʱ
	//printf("XX1:%f\n",(t4.QuadPart-t3.QuadPart)*1.0/tc.QuadPart);//��ӡ�˺�����ʱ
	return 0;
}

#endif




//#include <windows.h>
//#include <stdio.h>
//#include <stdlib.h>
//#define N 4
//
//int (*Queen)[N] = new int[100][N];
//int myCount=-1;
////��ʼ����ά����
//void InitMatrix(int L[][N])
//{
//	int i,j;
//	for(i=0; i<N; i++)
//		for(j=0; j<N; j++)
//			L[i][j] = 0;
//}
////�����ס������ϵĸ�
//void Mi(int L[][N], int row, int col)
//{
//	int i,j;
//	for(i=0; i<N; i++)
//	{
//		L[row][i] = -1;//ͬ��
//		L[i][col] = -1;//ͬ��
//	}
//	//��б��	//��б��
//	for(i=0; i<N; i++)
//		for(j=0; j<N; j++)
//			if((i-row == col-j)||(i-row == j-col))
//				L[i][j] = -1;
//}
////�Ϸ����жϡ�����rowĳ������֮ǰ�����бȽϣ�Q[]�洢֮ǰ�кϷ���λ��
//int isLegal(int L[], int row, int pos)
//{
//	for(int i=0; i<row; i++)
//	{
//		if(pos == L[i])return 0;
//		if(L[i] - pos == i - row)return 0;//���Խ���
//		if(L[i] - pos == row - i)return 0;//б�Խ���
//	}
//	return 1;
//}
////int isLegal(int Q[], int row, int col)
////{
////	int i;
////	if(row == 0)return 1;
////	for(i=0; i<row; i++)
////		if(col==Q[i] || col-i == row-Q[i] || i-col == row-Q[i])
////			return 0;
////	return 1;
////	
////}
////��������_������н�����ά����ĳ��
//void saveQueen(int Queen[][N], int cnt, int L[])
//{	
//	for(int i=0; i<N; i++)
//		Queen[cnt][i] = L[i];	
//}
////�̶�һ�����Ӻ󣬲��Ҹ�״̬�µ����п��ܣ����ؽ����
//int findQueen(int row, int col)
//{
//	int L[N][N], i,i0, i1, i2, i3;
//	int Q[N],count=-1;
//	for(i=0; i<N; i++)
//		Q[i] = 0;
//	InitMatrix(L);//��ʼ������
//	Mi(L, row, col);//�����ס������ϵĸ�
//	Q[row] = col;//��¼ָ������
//	//��ÿ����ȡһ�����ø񣬲��ж��Ƿ�Ϊ���н�
//
//		for(i0=0; i0<N; i0++)/*��0��*/
//			if(row==0)goto Layer1;
//			else
//				if(L[0][i0] != -1)//���ӿɷ�λ��
//				{
//					Q[0] = i0;
//					if(row==1)goto Layer2;
//					else
//			Layer1:		for(i1=0; i1<N; i1++)/*��1��*/
//							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
//							{
//								Q[1] = i1;
//								if(row==2)goto Layer3;
//								else
//			Layer2:					for(i2=0; i2<N; i2++)/*��2��*/
//										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
//										{
//											Q[2] = i2;
//											if(row==3)goto Layer4;
//											else
//			Layer3:								for(i3=0; i3<N; i3++)/*��3��*/
//													if(L[3][i3] != -1 && isLegal(Q, 3, i3))
//													{
//														Q[3] = i3;
//			Layer4:											//����Ϊ���н�
//														count++;
//														myCount++;
//														saveQueen(Queen,myCount,Q);
//														for(i=0; i<N; i++)
//															Q[i] = 0;
//													}
//										}
//							}
//			}	
//	return count+1;
//}
////��ӡ�ʺ�
//void showQueen(int Queen[][N], int count)
//{
//	for(int i=0; i<=count; i++)
//	{
//		printf("��%d���⣺ ",i+1);
//		for(int j=0; j<N; j++)
//			printf("(%d,%d)",j,Queen[i][j]);
//		printf("\n");
//	}
//}
////�������
////void Allocate()
////{
////	int i,j;
////	//������ÿ����Ϊһ������
////	for(i=0; i<N; i++)
////		for(j=0; j<N; j++)
////			findQueen(i, j);
////	
////}
//int main()
//{
//	int i=0,cnt,sum=0;
//
//			/*ʱ���ʱ����*/
//	LARGE_INTEGER t1,t2,tc;//����ʱ�����
//	QueryPerformanceFrequency(&tc);//
//	QueryPerformanceCounter(&t1);//
//	for(i=0; i<N; i++)
//	{
//		cnt = findQueen(0,i);
//		sum+=cnt;
//	}
//	QueryPerformanceCounter(&t2);//����ʱ��
//	printf("\n�ܹ�%d����",myCount+1);
//
//	printf("Use Time:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//��ӡ��ʱ
//	//showQueen(Queen, myCount);
//	return 0;
//}