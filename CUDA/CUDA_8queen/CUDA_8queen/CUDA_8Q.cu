
#include <hip/hip_runtime.h>
#if 0


/*
*N皇后，利用CUDA将传统的串行算法——盲目穷举，并行化
*
*
*/
#include "E:\VisualStudio\CUDA\common\book.h"
#include <windows.h>
#include <stdio.h>
#include <stdlib.h>

#define N 8

#define MAXSIZE 100
//int (*Queen)[N] = new int[INT_MAX/100][N];
//int (*Queen)[N] = (int (*)[N])malloc(MAXSIZE*N*sizeof(int));

//初始化二维数组
__device__ void InitMatrix(int L[][N])
{
	int i,j;
	for(i=0; i<N; i++)
		for(j=0; j<N; j++)
			L[i][j] = 0;
}
//处理【米】字线上的格
__device__ void Mi(int L[][N], int row, int col)
{
	int i,j;
	for(i=0; i<N; i++)
	{
		L[row][i] = -1;//同行
		L[i][col] = -1;//同列
	}
	//正斜线	//反斜线
	for(i=0; i<N; i++)
		for(j=0; j<N; j++)
			if((i-row == col-j)||(i-row == j-col))
				L[i][j] = -1;
}
//合法性判断——该row某列与其之前所有行比较，Q[]存储之前行合法列位置
__device__ int isLegal(int L[], int row, int pos )
{
	for(int i=0; i<row; i++)
	{
		if(L[i] == -1)return 0;
		if(pos == L[i])return 0;
		if(L[i] - pos == i - row)return 0;//正对角线
		if(L[i] - pos == row - i)return 0;//斜对角线
	}
	
	return 1;
}
//int isLegal(int Q[], int row, int col)
//{
//	int i;
//	if(row == 0)return 1;
//	for(i=0; i<row; i++)
//		if(col==Q[i] || col-i == row-Q[i] || i-col == row-Q[i])
//			return 0;
//	return 1;
//	
//}
//保存数组_保存可行解至二维数组某行
//__device__ void saveQueen(int Queen[][N], int cnt, int L[])
//{	
//	for(int i=0; i<N; i++)
//		Queen[cnt][i] = L[i];	
//}
__device__ static int myCount = 0;//统计解个数，没用上

//固定一个棋子后，查找该状态下的所有可能，返回解个数
/***************************************** 4皇后*****************************************/
//__device__ int findQueen4(int row, int col)//row为0，col为threadIdx
//{
//	//int i,j,flag;
//	//int L[N][N];
//	//int temp[N];
//	///*棋盘初始化*/
//	////正斜线	//反斜线
//	//for(i=0; i<N; i++)
//	//	for(j=0; j<N; j++)
//	//		if((i-row == col-j)||(i-row == j-col))
//	//			L[i][j] = -1;
//	//		else
//	//			L[i][j]=0;
//
//	//for(i=0; i<N; i++)
//	//{
//	//	L[row][i] = -1;//同行
//	//	L[i][col] = -1;//同列
//	//}
//	int cnt = 0;
//	int L[N] = {0};
//	L[row] = col;
//	for(int i1=0; i1<N; i1++)/*第1层*/
//	{
//		//if(L[1][i1]==-1)continue;		
//		if(isLegal(L, 1, i1 ))
//		{
//			L[1] = i1;
//			for(int i2=0; i2<N; i2++)/*第2层*/
//			{
//				
//				//if(L[2][i2]==-1)continue;
//				
//				if(isLegal(L, 2, i2 ))
//				{
//					L[2] = i2;
//					for(int i3=0; i3<N; i3++)/*第3层*/
//					{
//						
//						//if(L[3][i3]==-1)continue;
//						if(isLegal(L, 3, i3 ))
//						{
//							L[3] = i3;
//							cnt++;
//							myCount++;
//						}
//			
//					}
//				}
//			}
//		}
//	}
//				
//	return cnt;
//}
__device__ int findQueen4(int row, int col)
{

	int  i,i0, i1, i2, i3;
	int cnt = 0;
	int Q[N] = {0};
	int L[N][N] = {0};
	//for(i=0; i<N; i++)
	//	Q[i] = 0;
	//InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(row==3)goto Layer4;
												else
													if(L[3][i3] != -1 && isLegal(Q, 3, i3))
													{
														Q[3] = i3;
		Layer4:											//到此为可行解
														cnt++;
														//myCount++;
														//saveQueen(Queen,myCount,Q);
													}
										}
							}
			}	
	return cnt;
}
//***************************************** 5皇后*****************************************/
__device__ int findQueen5(int row, int col)
{
	int L[N][N] = {0};
	int i,i0, i1, i2, i3, i4;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
																//到此为可行解
																cnt++;
																//myCount++;
																//saveQueen(Queen,myCount,Q);
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 6皇后*****************************************/
__device__ int findQueen6(int row, int col)
{
	int L[N][N] = {0};
	int i,i0, i1, i2, i3, i4, i5;
	int Q[N] = {0};
	int cnt = 0;

	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																			//到此为可行解
																			cnt++;
																			//myCount++;
																			//saveQueen(Queen,myCount,Q);
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 7皇后*****************************************/
__device__ int findQueen7(int row, int col )
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																					//到此为可行解
																					cnt++;
																					//myCount++;
																					//saveQueen(Queen,myCount,Q);
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 8皇后*****************************************/
__device__ int findQueen8(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																							//到此为可行解
																							cnt++;
																							//myCount++;
																							//saveQueen(Queen,myCount,Q);
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
///***************************************** 9皇后*****************************************/
__device__ int findQueen9(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8;
	int Q[N] = {0} ;
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*第8层*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								//到此为可行解
																								cnt++;
																								//myCount++;
																								//saveQueen(Queen,myCount,Q);
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt ;
}
//***************************************** 10皇后*****************************************/
__device__ int findQueen10(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*第8层*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*第9层*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										//到此为可行解
																										cnt++;
																										//myCount++;
																										//saveQueen(Queen,myCount,Q);
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 11皇后*****************************************/
__device__ int findQueen11(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*第8层*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*第9层*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*第10层*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												//到此为可行解
																												cnt++;
																												//myCount++;
																												//saveQueen(Queen,myCount,Q);
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 12皇后*****************************************/
__device__ int findQueen12(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*第8层*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*第9层*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*第10层*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*第11层*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														//到此为可行解
																														cnt++;
																														//myCount++;
																														//saveQueen(Queen,myCount,Q);
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 13皇后*****************************************/
__device__ int findQueen13(int row, int col )
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*第8层*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*第9层*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*第10层*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*第11层*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*第12层*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																//到此为可行解
																																cnt++;
																																//myCount++;
																																//saveQueen(Queen,myCount,Q);
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 14皇后*****************************************/
__device__ int findQueen14(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*第8层*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*第9层*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*第10层*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*第11层*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*第12层*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*第13层*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		//到此为可行解
																																		cnt++;
																																		//myCount++;
																																		//saveQueen(Queen,myCount,Q);
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 15皇后*****************************************/
//int findQueen15(int row, int col)
//{
//	int L[N][N], i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14;
//	int Q[N],count=-1;
//	for(i=0; i<N; i++)
//		Q[i] = 0;
//	InitMatrix(L);//初始化棋盘
//	Mi(L, row, col);//处理【米】字线上的格
//	Q[row] = col;//记录指定棋子
//	//从每行提取一个可用格，并判断是否为可行解
//
//		//for(i0=0; i0<N; i0++)/*第0层*/
//			if(row==0)goto Layer1;
//			else
//				if(L[0][i0] != -1)//棋子可放位置
//				{
//					Q[0] = i0;
//
//		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
//						if(row==1)goto Layer2;
//						else
//							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
//							{
//								Q[1] = i1;
//
//		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
//									if(row==2)goto Layer3;
//									else
//										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
//										{
//											Q[2] = i2;
//
//		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
//												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
//												{
//													Q[3] = i3;
//													for(i4=0; i4<N; i4++)/*第4层*/
//													{
//														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
//														{
//															Q[4] = i4;
//															for(i5=0; i5<N; i5++)/*第5层*/
//															{																
//																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
//																	{
//																		Q[5] = i5;
//																		for(i6=0; i6<N; i6++)/*第6层*/
//																		{
//																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
//																			{
//																				Q[6] = i6;
//																				for(i7=0; i7<N; i7++)/*第7层*/
//																				{
//																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
//																					{
//																						Q[7] = i7;
//																						for(i8=0; i8<N; i8++)/*第8层*/
//																						{
//																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
//																							{
//																								Q[8] = i8;
//																								for(i9=0; i9<N; i9++)/*第9层*/
//																								{
//																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
//																									{
//																										Q[9] = i9;
//																										for(i10=0; i10<N; i10++)/*第10层*/
//																										{
//																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
//																											{
//																												Q[10] = i10;
//																												for(i11=0; i11<N; i11++)/*第11层*/
//																												{
//																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
//																													{
//																														Q[11] = i11;
//																														for(i12=0; i12<N; i12++)/*第12层*/
//																														{
//																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
//																															{
//																																Q[12] = i12;
//																																for(i13=0; i13<N; i13++)/*第13层*/
//																																{
//																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
//																																	{
//																																		Q[13] = i13;
//																																		for(i14=0; i14<N; i14++)/*第14层*/
//																																		{
//																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
//																																			{
//																																				Q[14] = i14;
//																																				//到此为可行解
//																																				count++;
//																																				myCount++;
//																																				saveQueen(Queen,myCount,Q);
//																																			}
//																																		}
//																																	}
//																																}
//																															}
//																														}
//																													}
//																												}
//																											}
//																										}
//																									}
//																								}
//																							}
//																						}
//																					}
//																				}
//																			}
//																		}
//																	}
//															}
//														}
//													}
//												}
//										}
//							}
//			}	
//	return count+1;
//}
__device__ int findQueen15(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*第8层*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*第9层*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*第10层*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*第11层*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*第12层*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*第13层*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		for(i14=0; i14<N; i14++)/*第14层*/
																																		{
																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
																																			{
																																				Q[14] = i14;
																																				//到此为可行解
																																				cnt++;
																																				//myCount++;
																																				//saveQueen(Queen,myCount,Q);
																																			}
																																		}
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 16皇后*****************************************/
__device__ int findQueen16(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14, i15;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*第8层*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*第9层*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*第10层*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*第11层*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*第12层*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*第13层*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		for(i14=0; i14<N; i14++)/*第14层*/
																																		{
																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
																																			{
																																				Q[14] = i14;
																																				for(i15=0; i15<N; i15++)/*第15层*/
																																				{
																																					if(L[15][i15] != -1 && isLegal(Q, 15, i15))
																																					{
																																						Q[15] = i15;
																																						//到此为可行解
																																						cnt++;
																																						//myCount++;
																																						//saveQueen(Queen,myCount,Q);
																																					}
																																				}
																																			}
																																		}
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt ;
}
//***************************************** 17皇后*****************************************/
__device__ int findQueen17(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14, i15, i16;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*第8层*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*第9层*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*第10层*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*第11层*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*第12层*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*第13层*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		for(i14=0; i14<N; i14++)/*第14层*/
																																		{
																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
																																			{
																																				Q[14] = i14;
																																				for(i15=0; i15<N; i15++)/*第15层*/
																																				{
																																					if(L[15][i15] != -1 && isLegal(Q, 15, i15))
																																					{
																																						Q[15] = i15;
																																						for(i16=0; i16<N; i16++)/*第16层*/
																																						{
																																							if(L[16][i16] != -1 && isLegal(Q, 16, i16))
																																							{
																																								Q[16] = i16;
																																								//到此为可行解
																																								cnt++;
																																								//myCount++;
																																								//saveQueen(Queen,myCount,Q);
																																							}
																																						}
																																					}
																																				}
																																			}
																																		}
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 18皇后*****************************************/
__device__ int findQueen18(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14, i15, i16, i17;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*第8层*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*第9层*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*第10层*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*第11层*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*第12层*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*第13层*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		for(i14=0; i14<N; i14++)/*第14层*/
																																		{
																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
																																			{
																																				Q[14] = i14;
																																				for(i15=0; i15<N; i15++)/*第15层*/
																																				{
																																					if(L[15][i15] != -1 && isLegal(Q, 15, i15))
																																					{
																																						Q[15] = i15;
																																						for(i16=0; i16<N; i16++)/*第16层*/
																																						{
																																							if(L[16][i16] != -1 && isLegal(Q, 16, i16))
																																							{
																																								Q[16] = i16;
																																								for(i17=0; i17<N; i17++)/*第17层*/
																																								{
																																									if(L[17][i17] != -1 && isLegal(Q, 17, i17))
																																									{
																																										Q[17] = i17;
																																										//到此为可行解
																																										cnt++;
																																										//myCount++;
																																										//saveQueen(Queen,myCount,Q);
																																									}
																																								}
																																							}
																																						}
																																					}
																																				}
																																			}
																																		}
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt ;
}
//***************************************** 19皇后*****************************************/
__device__ int findQueen19(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14, i15, i16, i17, i18;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*第8层*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*第9层*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*第10层*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*第11层*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*第12层*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*第13层*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		for(i14=0; i14<N; i14++)/*第14层*/
																																		{
																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
																																			{
																																				Q[14] = i14;
																																				for(i15=0; i15<N; i15++)/*第15层*/
																																				{
																																					if(L[15][i15] != -1 && isLegal(Q, 15, i15))
																																					{
																																						Q[15] = i15;
																																						for(i16=0; i16<N; i16++)/*第16层*/
																																						{
																																							if(L[16][i16] != -1 && isLegal(Q, 16, i16))
																																							{
																																								Q[16] = i16;
																																								for(i17=0; i17<N; i17++)/*第17层*/
																																								{
																																									if(L[17][i17] != -1 && isLegal(Q, 17, i17))
																																									{
																																										Q[17] = i17;
																																										for(i18=0; i18<N; i18++)/*第18层*/
																																										{
																																											if(L[18][i18] != -1 && isLegal(Q, 18, i18))
																																											{
																																												Q[18] = i18;
																																												//到此为可行解
																																												cnt++;
																																												//myCount++;
																																												//saveQueen(Queen,myCount,Q);
																																											}
																																										}
																																									}
																																								}
																																							}
																																						}
																																					}
																																				}
																																			}
																																		}
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}
//***************************************** 20皇后*****************************************/
__device__ int findQueen20(int row, int col)
{
	int L[N][N];
	int i,i0, i1, i2, i3, i4, i5, i6, i7, i8, i9, i10, i11, i12, i13, i14, i15, i16, i17, i18, i19;
	int Q[N] = {0};
	int cnt = 0;
	InitMatrix(L);//初始化棋盘
	Mi(L, row, col);//处理【米】字线上的格
	Q[row] = col;//记录指定棋子
	//从每行提取一个可用格，并判断是否为可行解

		//for(i0=0; i0<N; i0++)/*第0层*/
			if(row==0)goto Layer1;
			else
				if(L[0][i0] != -1)//棋子可放位置
				{
					Q[0] = i0;

		Layer1:		for(i1=0; i1<N; i1++)/*第1层*/      /*不该循环，却循环了4次，得到4个相同结果*/
						if(row==1)goto Layer2;
						else
							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
							{
								Q[1] = i1;

		Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
									if(row==2)goto Layer3;
									else
										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
										{
											Q[2] = i2;

		Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
												if(L[3][i3] != -1 && isLegal(Q, 3, i3))
												{
													Q[3] = i3;
													for(i4=0; i4<N; i4++)/*第4层*/
													{
														if(L[4][i4] != -1 && isLegal(Q, 4, i4))
														{
															Q[4] = i4;
															for(i5=0; i5<N; i5++)/*第5层*/
															{																
																if(L[5][i5] != -1 && isLegal(Q, 5, i5))
																	{
																		Q[5] = i5;
																		for(i6=0; i6<N; i6++)/*第6层*/
																		{
																			if(L[6][i6] != -1 && isLegal(Q, 6, i6))
																			{
																				Q[6] = i6;
																				for(i7=0; i7<N; i7++)/*第7层*/
																				{
																					if(L[7][i7] != -1 && isLegal(Q, 7, i7))
																					{
																						Q[7] = i7;
																						for(i8=0; i8<N; i8++)/*第8层*/
																						{
																							if(L[8][i8] != -1 && isLegal(Q, 8, i8))
																							{
																								Q[8] = i8;
																								for(i9=0; i9<N; i9++)/*第9层*/
																								{
																									if(L[9][i9] != -1 && isLegal(Q, 9, i9))
																									{
																										Q[9] = i9;
																										for(i10=0; i10<N; i10++)/*第10层*/
																										{
																											if(L[10][i10] != -1 && isLegal(Q, 10, i10))
																											{
																												Q[10] = i10;
																												for(i11=0; i11<N; i11++)/*第11层*/
																												{
																													if(L[11][i11] != -1 && isLegal(Q, 11, i11))
																													{
																														Q[11] = i11;
																														for(i12=0; i12<N; i12++)/*第12层*/
																														{
																															if(L[12][i12] != -1 && isLegal(Q, 12, i12))
																															{
																																Q[12] = i12;
																																for(i13=0; i13<N; i13++)/*第13层*/
																																{
																																	if(L[13][i13] != -1 && isLegal(Q, 13, i13))
																																	{
																																		Q[13] = i13;
																																		for(i14=0; i14<N; i14++)/*第14层*/
																																		{
																																			if(L[14][i14] != -1 && isLegal(Q, 14, i14))
																																			{
																																				Q[14] = i14;
																																				for(i15=0; i15<N; i15++)/*第15层*/
																																				{
																																					if(L[15][i15] != -1 && isLegal(Q, 15, i15))
																																					{
																																						Q[15] = i15;
																																						for(i16=0; i16<N; i16++)/*第16层*/
																																						{
																																							if(L[16][i16] != -1 && isLegal(Q, 16, i16))
																																							{
																																								Q[16] = i16;
																																								for(i17=0; i17<N; i17++)/*第17层*/
																																								{
																																									if(L[17][i17] != -1 && isLegal(Q, 17, i17))
																																									{
																																										Q[17] = i17;
																																										for(i18=0; i18<N; i18++)/*第18层*/
																																										{
																																											if(L[18][i18] != -1 && isLegal(Q, 18, i18))
																																											{
																																												Q[18] = i18;
																																												for(i19=0; i19<N; i19++)/*第19层*/
																																												{
																																													if(L[19][i19] != -1 && isLegal(Q, 19, i19))
																																													{
																																														Q[19] = i19;
																																														//到此为可行解
																																														cnt++;
																																														//myCount++;
																																														//saveQueen(Queen,myCount,Q);
																																													}
																																												}
																																											}
																																										}
																																									}
																																								}
																																							}
																																						}
																																					}
																																				}
																																			}
																																		}
																																	}
																																}
																															}
																														}
																													}
																												}
																											}
																										}
																									}
																								}
																							}
																						}
																					}
																				}
																			}
																		}
																	}
															}
														}
													}
												}
										}
							}
			}	
	return cnt;
}

//打印皇后
void showQueen(int Queen[][N], int count)
{
	for(int i=0; i<=count; i++)
	{
		printf("第%d个解： ",i+1);
		for(int j=0; j<N; j++)
			printf("(%d,%d)",j,Queen[i][j]);
		printf("\n");
	}
}

//任务分配
__global__ void kernel4(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen4(0, i);
			
}
__global__ void kernel5(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen5(0, i);
			
}
__global__ void kernel6(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen6(0, i);
			
}
__global__ void kernel7(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen7(0, i);
			
}
__global__ void kernel8(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen8(0, i);
			
}
__global__ void kernel9(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen9(0, i);
			
}
__global__ void kernel10(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen10(0, i);
			
}
__global__ void kernel11(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen11(0, i);
			
}
__global__ void kernel12(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen12(0, i);
			
}
__global__ void kernel13(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen13(0, i);
			
}
__global__ void kernel14(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen14(0, i);
			
}
__global__ void kernel15(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen15(0, i);
			
}
__global__ void kernel16(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen16(0, i);
			
}
__global__ void kernel17(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen17(0, i);
			
}
__global__ void kernel18(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen18(0, i);
			
}
__global__ void kernel19(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen19(0, i);
			
}
__global__ void kernel20(int* dev_c)
{
	int i = blockIdx.x;
	//int i=threadIdx.x;
	if(i < N)
		*(dev_c+i) = findQueen20(0, i);
			
}
int main()
{
	LARGE_INTEGER t1,t2,tc,t3,t4;//定义时间变量
	QueryPerformanceFrequency(&tc);
	QueryPerformanceCounter(&t1);//总执行时间start
	int i=0,sum=0;
	//int* dev_row;
	//int* dev_col;
	//int count[N] = {0};
	int* count = (int*)malloc(N*sizeof(int));
	for(i=0; i<N; i++)
		*(count+i) = 0;
	int* dev_c;
	//HANDLE_ERROR(cudaMalloc((void**)&dev_row,sizeof(int)));//行
	//HANDLE_ERROR(cudaMalloc((void**)&dev_col,sizeof(int)));//列
	HANDLE_ERROR(cudaMalloc((void**)&dev_c,N*sizeof(int)));
    HANDLE_ERROR( cudaMemcpy(dev_c, count,
                              N*sizeof(int),
                              cudaMemcpyHostToDevice) );

	QueryPerformanceCounter(&t2);//核函数开始时间
	switch(N)
	{
	case 4:
		//Allocate<<<1,N>>>(dev_c);
		kernel4<<<N,1>>>(dev_c);
		break;
	case 5:
		kernel5<<<N,1>>>(dev_c);
		break;
	case 6:
		kernel6<<<N,1>>>(dev_c);
		break;
	case 7:
		kernel7<<<N,1>>>(dev_c);
		break;
	case 8:
		kernel8<<<N,1>>>(dev_c);
		break;
	case 9:
		kernel9<<<N,1>>>(dev_c);
		break;
	case 10:
		kernel10<<<N,1>>>(dev_c);
		break;
	case 11:
		kernel11<<<N,1>>>(dev_c);
		break;
	case 12:
		kernel12<<<N,1>>>(dev_c);
		break;
	case 13:
		kernel13<<<N,1>>>(dev_c);
		break;
	case 14:
		kernel14<<<N,1>>>(dev_c);
	case 15:
		kernel15<<<N,1>>>(dev_c);
	case 16:
		kernel16<<<N,1>>>(dev_c);
	case 17:
		kernel17<<<N,1>>>(dev_c);
		break;
	case 18:
		kernel18<<<N,1>>>(dev_c);
		break;
	case 19:
		kernel19<<<N,1>>>(dev_c);
		break;
	case 20:
		kernel20<<<N,1>>>(dev_c);
			

		default:
			;
	
	}
	QueryPerformanceCounter(&t3);//核函数结束时间
	HANDLE_ERROR(cudaMemcpy(count,dev_c,N*sizeof(int),cudaMemcpyDeviceToHost));

	for(i=0; i<N; i++ )
		sum += count[i];
	//printf("\n总共%d个解\n", sum2);
	printf("\n总共%d个解\n", sum);
	QueryPerformanceCounter(&t4);//结束时间
	printf("Use Time:%f\n",(t4.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//打印耗时
	printf("Kernel Use Time:%f\n",(t3.QuadPart-t2.QuadPart)*1.0/tc.QuadPart);//打印核函数耗时

	//printf("XX1:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//打印核函数耗时
	//printf("XX1:%f\n",(t4.QuadPart-t3.QuadPart)*1.0/tc.QuadPart);//打印核函数耗时
	return 0;
}

#endif




//#include <windows.h>
//#include <stdio.h>
//#include <stdlib.h>
//#define N 4
//
//int (*Queen)[N] = new int[100][N];
//int myCount=-1;
////初始化二维数组
//void InitMatrix(int L[][N])
//{
//	int i,j;
//	for(i=0; i<N; i++)
//		for(j=0; j<N; j++)
//			L[i][j] = 0;
//}
////处理【米】字线上的格
//void Mi(int L[][N], int row, int col)
//{
//	int i,j;
//	for(i=0; i<N; i++)
//	{
//		L[row][i] = -1;//同行
//		L[i][col] = -1;//同列
//	}
//	//正斜线	//反斜线
//	for(i=0; i<N; i++)
//		for(j=0; j<N; j++)
//			if((i-row == col-j)||(i-row == j-col))
//				L[i][j] = -1;
//}
////合法性判断——该row某列与其之前所有行比较，Q[]存储之前行合法列位置
//int isLegal(int L[], int row, int pos)
//{
//	for(int i=0; i<row; i++)
//	{
//		if(pos == L[i])return 0;
//		if(L[i] - pos == i - row)return 0;//正对角线
//		if(L[i] - pos == row - i)return 0;//斜对角线
//	}
//	return 1;
//}
////int isLegal(int Q[], int row, int col)
////{
////	int i;
////	if(row == 0)return 1;
////	for(i=0; i<row; i++)
////		if(col==Q[i] || col-i == row-Q[i] || i-col == row-Q[i])
////			return 0;
////	return 1;
////	
////}
////保存数组_保存可行解至二维数组某行
//void saveQueen(int Queen[][N], int cnt, int L[])
//{	
//	for(int i=0; i<N; i++)
//		Queen[cnt][i] = L[i];	
//}
////固定一个棋子后，查找该状态下的所有可能，返回解个数
//int findQueen(int row, int col)
//{
//	int L[N][N], i,i0, i1, i2, i3;
//	int Q[N],count=-1;
//	for(i=0; i<N; i++)
//		Q[i] = 0;
//	InitMatrix(L);//初始化棋盘
//	Mi(L, row, col);//处理【米】字线上的格
//	Q[row] = col;//记录指定棋子
//	//从每行提取一个可用格，并判断是否为可行解
//
//		for(i0=0; i0<N; i0++)/*第0层*/
//			if(row==0)goto Layer1;
//			else
//				if(L[0][i0] != -1)//棋子可放位置
//				{
//					Q[0] = i0;
//					if(row==1)goto Layer2;
//					else
//			Layer1:		for(i1=0; i1<N; i1++)/*第1层*/
//							if(L[1][i1] != -1 && isLegal(Q, 1, i1))
//							{
//								Q[1] = i1;
//								if(row==2)goto Layer3;
//								else
//			Layer2:					for(i2=0; i2<N; i2++)/*第2层*/
//										if(L[2][i2] != -1 && isLegal(Q, 2, i2))
//										{
//											Q[2] = i2;
//											if(row==3)goto Layer4;
//											else
//			Layer3:								for(i3=0; i3<N; i3++)/*第3层*/
//													if(L[3][i3] != -1 && isLegal(Q, 3, i3))
//													{
//														Q[3] = i3;
//			Layer4:											//到此为可行解
//														count++;
//														myCount++;
//														saveQueen(Queen,myCount,Q);
//														for(i=0; i<N; i++)
//															Q[i] = 0;
//													}
//										}
//							}
//			}	
//	return count+1;
//}
////打印皇后
//void showQueen(int Queen[][N], int count)
//{
//	for(int i=0; i<=count; i++)
//	{
//		printf("第%d个解： ",i+1);
//		for(int j=0; j<N; j++)
//			printf("(%d,%d)",j,Queen[i][j]);
//		printf("\n");
//	}
//}
////任务分配
////void Allocate()
////{
////	int i,j;
////	//棋盘上每个格为一个任务
////	for(i=0; i<N; i++)
////		for(j=0; j<N; j++)
////			findQueen(i, j);
////	
////}
//int main()
//{
//	int i=0,cnt,sum=0;
//
//			/*时间计时函数*/
//	LARGE_INTEGER t1,t2,tc;//定义时间变量
//	QueryPerformanceFrequency(&tc);//
//	QueryPerformanceCounter(&t1);//
//	for(i=0; i<N; i++)
//	{
//		cnt = findQueen(0,i);
//		sum+=cnt;
//	}
//	QueryPerformanceCounter(&t2);//结束时间
//	printf("\n总共%d个解",myCount+1);
//
//	printf("Use Time:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//打印耗时
//	//showQueen(Queen, myCount);
//	return 0;
//}