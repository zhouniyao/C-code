#include "E:\VisualStudio\CUDA\CUDAColorToGray\CUDAColorToGray\bmploader_zhang.h"
#include"E:\VisualStudio\CUDA\CUDAColorToGray\common/cpu_bitmap.h"
#include<iostream>
#define WIDTH 640
#define	HEIGHT 480

void fade( unsigned char *dst ,unsigned char *ptr)//��ɫ
{
	int i = 0;
	 for (int y=0; y<HEIGHT; y++) {
        for (int x=0; x<WIDTH;  x++) {
            int offset = x + y * WIDTH;
			
		//Gray = R*0.299 + G*0.587 + B*0.114
            //ptr[offset*4 + 0] = ptr[offset*4 + 0]*0.299 + ptr[offset*4 + 1] *0.587 + ptr[offset*4 + 2] * 0.114 ;
            //ptr[offset*4 + 1] = ptr[offset*4 + 0]*0.299 + ptr[offset*4 + 1] *0.587 + ptr[offset*4 + 2] * 0.114 ;
            //ptr[offset*4 + 2] = ptr[offset*4 + 0]*0.299 + ptr[offset*4 + 1] *0.587 + ptr[offset*4 + 2] * 0.114 ;
			dst[i] = ptr[offset*4 + 0]*0.299 + ptr[offset*4 + 1] *0.587 + ptr[offset*4 + 2] * 0.114;
			i++;
		}
	 }
}

//�ۼ����ػҶ�ֵ0-255
void addgray(int Cpixels[], unsigned char *ptr, int nCount){
	for(int i=0; n<nCount; i++)
	{
		Cpixels[ptr[i]]++;
	}
}
int main()
{
	unsigned char *dst1;

	
	int w,h;
	w=WIDTH;h=HEIGHT;	
	dst1=(unsigned char *)malloc(w*h*sizeof(char));
	char *name="nature_monte.bmp";


	LoadBMPFile(&dst1, &w,  &h, name);	

	CPUBitmap bitmap(w, h/4);
	fade(bitmap.pixels, dst1);
	int Cpixels[256] = malloc(256*sizeof(int));
	memset(0,Cpixels);//��0
	bitmap.display_and_exit(); 
	free(dst1);
}
