#include "hip/hip_runtime.h"
#if 1
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */
/*
julia��������ĳ���������㺯�������е㹹�ɵı߽硣���ں�������������ȡֵ�����ɵı߽罫�γ�һ�ֲ��������Ƭ��״��������ѧ������Ȥ����Ư������״֮һ��
����julia���㷨�ǳ��򵥣�Julia���Ļ����㷨��,ͨ��һ���򵥵ĵ�����ʽ�Ը�ƽ���еĵ���ֵ��
����ڼ���ĳ����ʱ��������ʽ�ļ������Ƿ�ɢ�ģ���ô�����Ͳ�����Julia���ϣ�
�෴������ڵ�����ʽ�м���õ���һϵ��ֵ��λ��ĳ���߽緶Χ֮�ڣ���ô����������Julia���ϡ�
������ʽΪ��

	Z(n+1)=Z(n)*Z(n)+C

����GPU��Julia�����㷨���£�

*/

#include<windows.h>
#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define DIM 500

struct hipComplex {
    float   r;
    float   i;
    hipComplex( float a, float b ) : r(a), i(b)  {}
    float magnitude2( void ) { return r * r + i * i; }//��Ա������ģ��ƽ��
    hipComplex operator*(const hipComplex& a) {          //��Ա�����������˷�������
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};


int julia( int x, int y ) 
//�������ܣ�������ת��Ϊ����
{ 
    const float scale = 1.5;                         //&scale �ȼ����Ŵ�ϵ������ֵԽС��Խ�Ŵ�
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);   //����������ת��Ϊ�����ռ������
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);


    hipComplex c(-0.8, 0.156);        //&���������г���C�Ķ���
    hipComplex a(jx, jy);             //����������Z�����ĳ�ֵ

    int i = 0;
    for (i=0; i<100; i++) {          //$$$����200�ε���(��������Խ�࣬ͼ��Խ��ϸ)
        a = a * a + c;
        if (a.magnitude2() >1000)   //ģ��ƽ������1000����ֵ���Ļ�����˵����ɢ��������julia��
            return 0;
    }

    return 1;                //Z(a)����julia�����򷵼�1
}

/*��һ������kernel����Ϊһ��__global__ ���͵ĺ������̸߳�ÿһά��
��С��ͼ��ÿһά��С����ȵģ���Щ�ڣ�0��0��
�ͣ�DIM-1��DIM-1��֮���ÿ�����ص㶼�ܻ��һ���߳̿�
�ڶ������õ���� ������ptr�е�����ƫ�ƣ����ƫ����ͨ����һ��
���ñ���girdDim������ģ������е��߳̿���˵��gidDim��һ��������
���������̸߳��һά�Ĵ�С.��ʾ���У�gridDim��ֵ��(DIM,DIM), ��ˣ�
�������������̸߳�Ŀ��ټ������������͵õ���ptr�е�Ψһ������
��ȡֵ��ΧΪ��DIM*DIM-1����
�������ж�ĳ�����Ƿ�����Julia���Ĵ��롣
*/
/*
__global__ void kernel(unsigned char *ptr)
{
   //��threadIdx/BlockIdxӳ������λ��
    int x=blockIdx.x;
    int y=blockIdx.y;
    int offset=x+y*gridDim.x; //�̸߳�һά�Ĵ�С���Ǹ�����

    //�������λ���ϵ�ֵ
    int juliaValue=julia(x,y);
 
     ptr[offset*4+0]=255*juliaValue;
     ptr[offset*4+1]=0;
     ptr[offset*4+2]=0;
     ptr[offset*4+3]=255;
}

*/
void kernel( unsigned char *ptr ){
    for (int y=0; y<DIM; y++) {
        for (int x=0; x<DIM; x++) {
            int offset = x + y * DIM;//��һά��ʾͼ���е����е�

            int juliaValue = julia( x, y );
            ptr[offset*4 + 0] = 255 * juliaValue;   //R
            ptr[offset*4 + 1] = 0;					//G
            ptr[offset*4 + 2] = 0;				    //B
            ptr[offset*4 + 3] = 255;                //͸����
        }
    }
 }

//ͨ�����߿ⴴ��һ����С���ʵ�λͼͼ����һ��ָ��λͼ���ݵ�ָ�봫�ݸ��˺�����
int main( void ) {
    CPUBitmap bitmap( DIM, DIM );  //����λͼ����
	unsigned char *ptr = bitmap.get_ptr();//���������豸�����ݵĸ�����

    kernel( ptr );
	bitmap.display_and_exit();
	return 0;
}
#endif
#if 0
	/*ʱ���ʱ����*/
	LARGE_INTEGER t1,t2,tc;//����ʱ�����
	QueryPerformanceFrequency(&tc);//
	QueryPerformanceCounter(&t1);//
	��������
	QueryPerformanceCounter(&t2);//
	printf("Use Time:%f\n",(t2.QuadPart-t1.QuadPart)*1.0/tc.QuadPart);//��ӡ��ʱ

#endif