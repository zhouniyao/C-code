
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#if 0
#include "cuda.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"


#define DIM 1000
#define PI 3.1415926535897932f


struct cuComplex {
	float   r;
	float   i;
	__device__ cuComplex(float a, float b) : r(a), i(b)  {}
	__device__ float magnitude2(void) {
		return r * r + i * i;
	}
	__device__ cuComplex operator*(const cuComplex& a) {
		return cuComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}
	__device__ cuComplex operator+(const cuComplex& a) {
		return cuComplex(r + a.r, i + a.i);
	}
};

__device__ int julia(int x, int y) {
	const float scale = 1.5;
	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

	cuComplex c(-0.8, 0.156);
	cuComplex a(jx, jy);

	int i = 0;
	for (i = 0; i<200; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}

	return 1;
}

//以上从julia_gpu搬来，没有改动

__global__ void kernel( unsigned char *ptr, int ticks ) {
	// map from blockIdx to pixel position
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;

	// now calculate the value at that position
	int juliaValue = julia(x, y);
	
	ptr[offset * 4 + 0] = (255-ticks*20)* juliaValue;       //变色的秘密所在。  每次调用和函数，ticks都自增。 ticks乘以20的目的是，让变色效果更明显。
	ptr[offset * 4 + 1] = (100+ticks*20) *juliaValue;
	ptr[offset * 4 + 2] = (25+ticks*20) * juliaValue;		//前三个ptr是RGB
	ptr[offset * 4 + 3] = 255;								//最后一个ptr是透明度alpha
}

struct DataBlock {                                  //让CPU和GPU的存储可以相互访问
    unsigned char   *dev_bitmap;
    CPUAnimBitmap  *bitmap;
};

void generate_frame(DataBlock *d, int ticks) {
	
   // dim3    blocks(DIM/16,DIM/16);
	dim3    grid(DIM, DIM);       
   // dim3    threads(16,16);
    //kernel<<<blocks,threads>>>( d->dev_bitmap, ticks );
	kernel << <grid, 1 >> >(d->dev_bitmap,ticks++);
	printf("%d", ticks);                //此句在控制台显示ticks的值，可以把它注释掉
    HANDLE_ERROR( cudaMemcpy( d->bitmap->get_ptr(),
                              d->dev_bitmap,
                              d->bitmap->image_size(),
                              cudaMemcpyDeviceToHost ) );

	/*HANDLE_ERROR(cudaMemcpy(bitmap.get_ptr(), dev_bitmap,
		bitmap.image_size(),
		cudaMemcpyDeviceToHost));*/
}

// clean up memory allocated on the GPU
void cleanup( DataBlock *d ) {
    HANDLE_ERROR( cudaFree( d->dev_bitmap ) ); 
	//HANDLE_ERROR(cudaFree(dev_bitmap));
}

int main( void ) {
    DataBlock   data;
    CPUAnimBitmap  bitmap( DIM, DIM, &data );
    data.bitmap = &bitmap;       ////////相互指，可以互相访问

    HANDLE_ERROR( cudaMalloc( (void**)&data.dev_bitmap,    //////data这个结构体里只有两个指针
                              bitmap.image_size() ) );    

    bitmap.anim_and_exit( (void (*)(void*,int))generate_frame,    /////void(*)函数指针，generate_frame 产生动画帧///julia换色
                            (void (*)(void*))cleanup );   /////cleanup 清除

	
}
#endif