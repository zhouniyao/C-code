
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#if 0
#include "cuda.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"


#define DIM 1000
#define PI 3.1415926535897932f


struct cuComplex {
	float   r;
	float   i;
	__device__ cuComplex(float a, float b) : r(a), i(b)  {}
	__device__ float magnitude2(void) {
		return r * r + i * i;
	}
	__device__ cuComplex operator*(const cuComplex& a) {
		return cuComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}
	__device__ cuComplex operator+(const cuComplex& a) {
		return cuComplex(r + a.r, i + a.i);
	}
};

__device__ int julia(int x, int y) {
	const float scale = 1.5;
	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

	cuComplex c(-0.8, 0.156);
	cuComplex a(jx, jy);

	int i = 0;
	for (i = 0; i<200; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}

	return 1;
}

//���ϴ�julia_gpu������û�иĶ�

__global__ void kernel( unsigned char *ptr, int ticks ) {
	// map from blockIdx to pixel position
	int x = blockIdx.x;
	int y = blockIdx.y;
	int offset = x + y * gridDim.x;

	// now calculate the value at that position
	int juliaValue = julia(x, y);
	
	ptr[offset * 4 + 0] = (255-ticks*20)* juliaValue;       //��ɫ���������ڡ�  ÿ�ε��úͺ�����ticks�������� ticks����20��Ŀ���ǣ��ñ�ɫЧ�������ԡ�
	ptr[offset * 4 + 1] = (100+ticks*20) *juliaValue;
	ptr[offset * 4 + 2] = (25+ticks*20) * juliaValue;		//ǰ����ptr��RGB
	ptr[offset * 4 + 3] = 255;								//���һ��ptr��͸����alpha
}

struct DataBlock {                                  //��CPU��GPU�Ĵ洢�����໥����
    unsigned char   *dev_bitmap;
    CPUAnimBitmap  *bitmap;
};

void generate_frame(DataBlock *d, int ticks) {
	
   // dim3    blocks(DIM/16,DIM/16);
	dim3    grid(DIM, DIM);       
   // dim3    threads(16,16);
    //kernel<<<blocks,threads>>>( d->dev_bitmap, ticks );
	kernel << <grid, 1 >> >(d->dev_bitmap,ticks++);
	printf("%d", ticks);                //�˾��ڿ���̨��ʾticks��ֵ�����԰���ע�͵�
    HANDLE_ERROR( cudaMemcpy( d->bitmap->get_ptr(),
                              d->dev_bitmap,
                              d->bitmap->image_size(),
                              cudaMemcpyDeviceToHost ) );

	/*HANDLE_ERROR(cudaMemcpy(bitmap.get_ptr(), dev_bitmap,
		bitmap.image_size(),
		cudaMemcpyDeviceToHost));*/
}

// clean up memory allocated on the GPU
void cleanup( DataBlock *d ) {
    HANDLE_ERROR( cudaFree( d->dev_bitmap ) ); 
	//HANDLE_ERROR(cudaFree(dev_bitmap));
}

int main( void ) {
    DataBlock   data;
    CPUAnimBitmap  bitmap( DIM, DIM, &data );
    data.bitmap = &bitmap;       ////////�໥ָ�����Ի������

    HANDLE_ERROR( cudaMalloc( (void**)&data.dev_bitmap,    //////data����ṹ����ֻ������ָ��
                              bitmap.image_size() ) );    

    bitmap.anim_and_exit( (void (*)(void*,int))generate_frame,    /////void(*)����ָ�룬generate_frame ��������֡///julia��ɫ
                            (void (*)(void*))cleanup );   /////cleanup ���

	
}
#endif