
#include <hip/hip_runtime.h>
/*
* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
*
* NVIDIA Corporation and its licensors retain all intellectual property and
* proprietary rights in and to this software and related documentation.
* Any use, reproduction, disclosure, or distribution of this software
* and related documentation without an express license agreement from
* NVIDIA Corporation is strictly prohibited.
*
* Please refer to the applicable NVIDIA end user license agreement (EULA)
* associated with this source code for terms and conditions that govern
* your use of this NVIDIA software.
*
*/
#if 0

#include "../common/book.h"
#include "cpu_bitmap_SU.h"    //引用cpu_bitmap的副本

#define DIM 1000



struct cuComplex {
	float   r;
	float   i;
	cuComplex(float a, float b) : r(a), i(b)  {}
	float magnitude2(void) { return r * r + i * i; }
	cuComplex operator*(const cuComplex& a) {
		return cuComplex(r*a.r - i*a.i, i*a.r + r*a.i);
	}
	cuComplex operator+(const cuComplex& a) {
		return cuComplex(r + a.r, i + a.i);
	}
};

int julia(int x, int y) {
	const float scale = 1.5;
	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

	cuComplex c(-0.8, 0.156);
	cuComplex a(jx, jy);

	int i = 0;
	for (i = 0; i<200; i++) {
		a = a * a + c;
		if (a.magnitude2() > 1000)
			return 0;
	}

	return 1;
}
///以上为朱丽叶级数相关，原模原样复制过来，不用改动


//DataBlock的作用是在各个函数之间传递图像的内存
struct DataBlock {
	unsigned char   *dev_bitmap;   //存储要涂色的区域
	CPUBitmap  *bitmap;
};


void kernel(unsigned char *ptr,int ticks){
	printf("kernel:%d\n",ticks);
	for (int y = 0; y<DIM; y++) {
		for (int x = 0; x<DIM; x++) {
			int offset = x + y * DIM;
			int juliaValue = julia(x, y);
			ptr[offset * 4 + 0] = (255-ticks*10) * juliaValue;  
			ptr[offset * 4 + 1] = (0 + ticks * 15)* juliaValue;
			ptr[offset * 4 + 2] = (100 + ticks * 25)* juliaValue;  //前三个ptr用来上色，为了让变色更明显，所以令ticks乘以一个值
			ptr[offset * 4 + 3] = 255;  //决定透明度
		}
	}
}


//不断变色
void generate_frame(DataBlock *d, int ticks) {
	unsigned char *ptr =  d->dev_bitmap;  //获取图像像素点的内存地址
	kernel(ptr, ticks);  //这儿是生成图像的核函数
}


void cleanup(DataBlock *d) {
	free(d->dev_bitmap);   
}
int main(void) {
	DataBlock data;
	CPUBitmap bitmap(DIM, DIM,&data);
	data.bitmap = &bitmap;
	data.dev_bitmap = bitmap.get_ptr();
	//unsigned char *ptr =bitmap.get_ptr();

	//kernel(ptr,1);
	//printf("frame:%p\n", generate_frame); 
	bitmap.display_and_exit((void(*)(void*, int))generate_frame,
		(void(*)(void*))cleanup);
}

#endif