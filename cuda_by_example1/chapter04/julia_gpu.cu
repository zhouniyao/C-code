
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */
#if 0
#include"windows.h"
#include "../common/book.h"
#include "../common/cpu_bitmap.h"

#define DIM 1000


struct cuComplex {
    float   r;
    float   i;
    __device__ cuComplex( float a, float b ) : r(a), i(b)  {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ cuComplex operator*(const cuComplex& a) {
        return cuComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ cuComplex operator+(const cuComplex& a) {
        return cuComplex(r+a.r, i+a.i);
    }
};

__device__ int julia( int x, int y ) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    cuComplex c(-0.8, 0.156);
    cuComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernel( unsigned char *ptr , int*  dev_ix, int*  dev_iy) {
    // map from blockIdx to pixel position
    //int x = blockIdx.x;
    //int y = blockIdx.y;
    //int offset = x + y * gridDim.x;//gridDim�̸߳�ÿһά�Ĵ�С
	*dev_ix = gridDim.x;//blockDim.x = 8;gridDim.x = 125
	*dev_iy = gridDim.y;//blockDim.y = 8;gridDim.y = 125

	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	int offset = x + y *gridDim.x * blockDim.x;
    // now calculate the value at that position
    int juliaValue = julia( x, y );
    ptr[offset*4 + 0] = 255 * juliaValue;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}

// globals needed by the update routine
struct DataBlock {
    unsigned char   *dev_bitmap;
};

int main( void ) {
    DataBlock   data;
    CPUBitmap bitmap( DIM, DIM, &data );
    unsigned char    *dev_bitmap;

	int ix,iy;
	int* dev_ix;
	int* dev_iy;
	cudaMalloc((void**)&dev_ix,sizeof(int));
	cudaMalloc((void**)&dev_iy,sizeof(int));
    HANDLE_ERROR( cudaMalloc( (void**)&dev_bitmap, bitmap.image_size() ) );
    data.dev_bitmap = dev_bitmap;

    dim3    grid(DIM/8,DIM/8);
	dim3    threads(8,8);
    kernel<<<grid,threads>>>( dev_bitmap , dev_ix, dev_iy);

    HANDLE_ERROR( cudaMemcpy( bitmap.get_ptr(), dev_bitmap,
                              bitmap.image_size(),
                              cudaMemcpyDeviceToHost ) );
    cudaMemcpy(&ix,dev_ix,sizeof(int),cudaMemcpyDeviceToHost );    
	cudaMemcpy(&iy,dev_iy,sizeof(int),cudaMemcpyDeviceToHost );   
    HANDLE_ERROR( cudaFree( dev_bitmap ) );

/*	printf("=================");
     printf("%d  %d",ix,iy);  */                       
    bitmap.display_and_exit();
}

#endif