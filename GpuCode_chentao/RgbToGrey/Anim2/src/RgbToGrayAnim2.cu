#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"

clock_t start_time;
int frame_count = 0;

__global__ void kernel(unsigned char *ptr, int ticks)
{
    // map from threadIdx/BlockIdx to pixel position
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    int ty = ticks / gridDim.x;
    int tx = ticks % gridDim.x;

    if ((int) sqrtf(
            (float) pow((float) (tx - x), 2.0f)
                    + (float) pow((float) ty - y, 2.0f)) < 10)
    {
        unsigned char old_red = ptr[offset * 4 + 0];
        unsigned char old_green = ptr[offset * 4 + 1];
        unsigned char old_blue = ptr[offset * 4 + 2];

        //Gray=(R*30+G*59+B*11)/100
        //unsigned int grey = (unsigned int)(old_red*30+old_green*59+old_blue*11+50)/100;
        unsigned int grey = (int)((old_red+old_green+old_blue)/3.0+0.5);
        //unsigned char grey = 50;
        ptr[offset * 4 + 0] = grey;
        ptr[offset * 4 + 1] = grey;
        ptr[offset * 4 + 2] = grey;
        ptr[offset * 4 + 3] = 255;
        //*(unsigned int *)(ptr+offset*4) = grey;
    }

}

struct DataBlock
{
    unsigned char *dev_bitmap;
    CPUAnimBitmap *bitmap;
};

void generate_frame(DataBlock *d, int ticks)
{
    int w = d->bitmap->width;
    int h = d->bitmap->height;
    dim3 blocks(w, h);
    kernel<<<blocks, 1>>>(d->dev_bitmap, ticks);

    HANDLE_ERROR(
            hipMemcpy(d->bitmap->get_ptr(), d->dev_bitmap,
                    d->bitmap->image_size(), hipMemcpyDeviceToHost));

    ++frame_count;
    if ((double) (clock() - start_time) / CLOCKS_PER_SEC > 1.0)
    {
        printf("the fps is:%d\n", frame_count);
        frame_count = 0;
        start_time = clock();
    }
}

// clean up memory allocated on the GPU
void cleanup(DataBlock *d)
{
    HANDLE_ERROR(hipFree(d->dev_bitmap));
}

int main(void)
{
    DataBlock data;
    start_time = clock();
    CPUAnimBitmap bitmap("/home/ct/Pictures/b16.bmp", &data);
    data.bitmap = &bitmap;
    HANDLE_ERROR(hipMalloc((void** )&data.dev_bitmap, bitmap.image_size()));
    HANDLE_ERROR(
            hipMemcpy(data.dev_bitmap, bitmap.get_ptr(), bitmap.image_size(),
                    hipMemcpyHostToDevice));
    bitmap.anim_and_exit((void (*)(void*, int))generate_frame, (void (*)(void*))cleanup );
}
