#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */


#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <vector>
using namespace std;
#define DIM 1000
int count = 0;

vector<int> JuliaDot_Vector;
vector<int> JuliaDotAlpha_Vector;
vector<int> JuliaDotStep_Vector;
bool GPU_FLAG = false;
bool ALPHA_FLAG = false;
clock_t start_time;
//#define PI 3.1415926535897932f

//__global__ void kernel( unsigned char *ptr, int ticks ) {
//    // map from threadIdx/BlockIdx to pixel position
//    int x = threadIdx.x + blockIdx.x * blockDim.x;
//    int y = threadIdx.y + blockIdx.y * blockDim.y;
//    int offset = x + y * blockDim.x * gridDim.x;
//
//    // now calculate the value at that position
//    float fx = x - DIM/2;
//    float fy = y - DIM/2;
//    float d = sqrtf( fx * fx + fy * fy );
//    unsigned char grey = (unsigned char)(128.0f + 127.0f *
//                                         cos(d/10.0f - ticks/7.0f) /
//                                         (d/10.0f + 1.0f));
//    ptr[offset*4 + 0] = grey;
//    ptr[offset*4 + 1] = grey;
//    ptr[offset*4 + 2] = grey;
//    ptr[offset*4 + 3] = 255;
//}
struct hipComplex {
    float   r;
    float   i;
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};
__device__ int julia( int x, int y ) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}
__global__ void kernel2( unsigned char *ptr ) {
    // map from blockIdx to pixel position
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // now calculate the value at that position
    int juliaValue = julia( x, y );
    ptr[offset*4 + 0] = juliaValue*255;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;

}
struct DataBlock {
    unsigned char   *dev_bitmap;
    CPUAnimBitmap  *bitmap;
};

void generate_frame( DataBlock *d, int ticks ) {
	count ++ ;
	if((double)(clock()-start_time) / CLOCKS_PER_SEC > 1.0 ){
		printf("fps:%d\n",count);
		start_time = clock();
		count = 0;
	}
    if(false == GPU_FLAG){
    	dim3    grid(DIM,DIM);
    	kernel2<<<grid,1>>>( d->dev_bitmap);
    	HANDLE_ERROR( hipMemcpy( d->bitmap->get_ptr(),
                              d->dev_bitmap,
                              d->bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );
    	GPU_FLAG = true;
    	for(int i=0;i<DIM*DIM;i++){
    		if( 255 == *(d->bitmap->get_ptr()+i*4) ){
    			JuliaDot_Vector.push_back(i);
    			JuliaDotAlpha_Vector.push_back(255);
    			JuliaDotStep_Vector.push_back(random()%15+1);
    		}
    	}
    }else{
    	for(int i=0;i<JuliaDot_Vector.size();i++){
    		*(d->bitmap->get_ptr()+JuliaDot_Vector[i]*4)=255;
    		*(d->bitmap->get_ptr()+JuliaDot_Vector[i]*4+1)=255;
    		*(d->bitmap->get_ptr()+JuliaDot_Vector[i]*4+2)=255;
    		*(d->bitmap->get_ptr()+JuliaDot_Vector[i]*4+3)=JuliaDotAlpha_Vector[i];
    		if(false == ALPHA_FLAG)
    		{
    			if(JuliaDotAlpha_Vector[i] > 0 ){
    				JuliaDotAlpha_Vector[i] -= JuliaDotStep_Vector[i];
    			}
    			else{
    				ALPHA_FLAG = true;
    			}
    		}else{
    			if(JuliaDotAlpha_Vector[i] < 255){
    				JuliaDotAlpha_Vector[i] += JuliaDotStep_Vector[i];
    			}
    			else{
    				ALPHA_FLAG = false;
    			}
    		}
    	}
    }
}
    // the next time not use GPU to calculate julia

// clean up memory allocated on the GPU
void cleanup( DataBlock *d ) {
    HANDLE_ERROR( hipFree( d->dev_bitmap ) );
}

int main( void ) {

    DataBlock   data;
    CPUAnimBitmap  bitmap( DIM, DIM, &data );
    data.bitmap = &bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&data.dev_bitmap,
                              bitmap.image_size() ) );
    start_time = clock();
    bitmap.anim_and_exit( (void (*)(void*,int))generate_frame,
                            (void (*)(void*))cleanup );
}
