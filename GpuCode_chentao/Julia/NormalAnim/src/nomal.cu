#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "../common/cpu_anim.h"
#include <stdio.h>
#include <time.h>
#define DIM 1000
int count = 0;
clock_t start_time;
//#define PI 3.1415926535897932f

//__global__ void kernel( unsigned char *ptr, int ticks ) {
//    // map from threadIdx/BlockIdx to pixel position
//    int x = threadIdx.x + blockIdx.x * blockDim.x;
//    int y = threadIdx.y + blockIdx.y * blockDim.y;
//    int offset = x + y * blockDim.x * gridDim.x;
//
//    // now calculate the value at that position
//    float fx = x - DIM/2;
//    float fy = y - DIM/2;
//    float d = sqrtf( fx * fx + fy * fy );
//    unsigned char grey = (unsigned char)(128.0f + 127.0f *
//                                         cos(d/10.0f - ticks/7.0f) /
//                                         (d/10.0f + 1.0f));
//    ptr[offset*4 + 0] = grey;
//    ptr[offset*4 + 1] = grey;
//    ptr[offset*4 + 2] = grey;
//    ptr[offset*4 + 3] = 255;
//}
struct hipComplex {
    float   r;
    float   i;
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};
__device__ int julia( int x, int y ) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}
__global__ void kernel2( unsigned char *ptr ,int rand) {
    // map from blockIdx to pixel position
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // now calculate the value at that position
    int juliaValue = julia( x, y );
    ptr[offset*4 + 0] = juliaValue*rand;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 0;
    ptr[offset*4 + 3] = 255;
}
struct DataBlock {
    unsigned char   *dev_bitmap;
    CPUAnimBitmap  *bitmap;
};

void generate_frame( DataBlock *d, int ticks ) {
	static int times = 1;
	times = times+10;
	times = times%255+1;

	count ++ ;
	if((double)(clock()-start_time) / CLOCKS_PER_SEC > 1.0 ){
		printf("fps:%d\n",count);
		start_time = clock();
		count = 0;
	}
    dim3    grid(DIM,DIM);
    kernel2<<<grid,1>>>( d->dev_bitmap ,times);
    HANDLE_ERROR( hipMemcpy( d->bitmap->get_ptr(),
                              d->dev_bitmap,
                              d->bitmap->image_size(),
                              hipMemcpyDeviceToHost ) );
}

// clean up memory allocated on the GPU
void cleanup( DataBlock *d ) {
    HANDLE_ERROR( hipFree( d->dev_bitmap ) ); 
}

int main( void ) {

    DataBlock   data;
    CPUAnimBitmap  bitmap( DIM, DIM, &data );
    data.bitmap = &bitmap;

    HANDLE_ERROR( hipMalloc( (void**)&data.dev_bitmap,
                              bitmap.image_size() ) );
    start_time = clock();
    bitmap.anim_and_exit( (void (*)(void*,int))generate_frame,
                            (void (*)(void*))cleanup );
}
